
// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include <ATen/native/transformers/cuda/flash_attn/flash_fwd_launch_template.h>
namespace pytorch_flash{

template<>
void run_mha_fwd_<cutlass::half_t, 160>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim160<cutlass::half_t>(params, stream);
}
} // namespace pytorch_flash
