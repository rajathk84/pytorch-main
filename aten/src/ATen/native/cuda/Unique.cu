#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch_v2.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/ThrustAllocator.h>

#include <c10/util/Load.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#else
#include <ATen/ops/_unique2_native.h>
#include <ATen/ops/_unique_native.h>
#include <ATen/ops/arange.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/unique_consecutive_native.h>
#include <ATen/ops/unique_dim_consecutive_native.h>
#include <ATen/ops/unique_dim_native.h>
#endif

#include <tuple>
#include <iterator>
#include <thrust/adjacent_difference.h>
#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>

#include <ATen/native/cuda/UniqueCub.cuh>

namespace at::native {

namespace {

template <
  typename policy_t, typename scalar_t,
  typename equal_t, typename not_equal_t
>
std::tuple<Tensor, Tensor, int64_t> compute_unique(
  const policy_t &policy,
  scalar_t *data,
  int64_t num_inp,
  const Tensor &sorted_indices,
  const bool return_inverse,
  const bool return_counts,
  TensorOptions options,
  equal_t equal,
  not_equal_t not_equal
) {
  // inverse indices
  Tensor inverse_indices;
  if (!return_inverse || num_inp == 0) {
    inverse_indices = at::empty({0}, options);
  } else {
    TORCH_CHECK(sorted_indices.defined(),
      "return_inverse is set to true, but sorted_indices is undefined. Send a bug report!");
    const int64_t *sorted_indices_ptr = sorted_indices.const_data_ptr<int64_t>();
    Tensor inv_loc = at::empty({num_inp}, options);
    inverse_indices = at::empty({num_inp}, options);
    int64_t* inv_loc_ptr = inv_loc.mutable_data_ptr<int64_t>();
    int64_t* inverse_indices_ptr = inverse_indices.mutable_data_ptr<int64_t>();
    thrust::adjacent_difference(policy, data, data + num_inp, inv_loc_ptr, not_equal);
    inv_loc[0] = 0;
    thrust::inclusive_scan(policy, inv_loc_ptr, inv_loc_ptr + num_inp, inv_loc_ptr);
    thrust::scatter(policy, inv_loc_ptr, inv_loc_ptr + num_inp, sorted_indices_ptr, inverse_indices_ptr);
  }

  // unique and count
  Tensor counts = at::empty({0}, options);
  int64_t num_out;
  if (!return_counts) {
    num_out = thrust::unique(policy, data, data + num_inp, equal) - data;
  } else {
    Tensor range = at::arange(0, num_inp + 1, options);
    int64_t *range_ptr = range.mutable_data_ptr<int64_t>();
    num_out = thrust::unique_by_key(policy, data, data + num_inp, range_ptr, equal).first - data;
    range[num_out] = num_inp;
    counts.resize_(num_out);
    int64_t* counts_ptr = counts.mutable_data_ptr<int64_t>();
    thrust::adjacent_difference(policy, range_ptr + 1, range_ptr + num_out + 1, counts_ptr);
  }

  AT_CUDA_CHECK(hipGetLastError());
  return std::tuple<Tensor, Tensor, int64_t>(inverse_indices, counts, num_out);
}

template <typename scalar_t>
std::tuple<Tensor, Tensor, Tensor> unique_dim_cuda_template(
  const Tensor& self,
  const int64_t dim,
  const bool consecutive,
  const bool return_inverse,
  const bool return_counts
) {

  /**
    * The idea for implementing this is basically the same as unique.
    * For unique_dim, we are taking the unique with respect to a index
    * tensor, but during the processes, we override the compare and equal
    * operator by checking the data underlying it instead. After the
    * algorithm, we would use index_select to map the resulting indices
    * to the result on the actual data.
    */

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::cuda::ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto sizes = self.sizes().vec();
  // check how many zero dimensions exist
  auto num_zero_dims = std::count(sizes.begin(), sizes.end(), 0);

  // tensor is not well formed as it has 0 sized dimensions
  if (self.size(dim) == 0){
    TORCH_CHECK(
        num_zero_dims == 1,
        "Number of zero sized dimensions is more than one, so unique cannot be applied ")
    Tensor output = at::empty(sizes, self.options());
    Tensor inverse_indices =
        at::empty({0}, self.options().dtype(kLong));
    Tensor counts = at::empty({0}, self.options().dtype(kLong));

    return std::make_tuple(output, inverse_indices, counts);
  }

  TORCH_CHECK(num_zero_dims == 0,
    "There are 0 sized dimensions, and they aren't selected, so unique cannot be applied");

  int64_t num_inp = self.size(dim);
  auto options = self.options().dtype(kLong);
  Tensor input_flat = self.moveaxis(dim, 0).contiguous().view({num_inp, -1});
  int64_t n = input_flat.size(1);
  const scalar_t *input_flat_ptr = input_flat.const_data_ptr<scalar_t>();

  Tensor indices = at::arange(0, num_inp, options);
  int64_t *indices_data = indices.mutable_data_ptr<int64_t>();
  if (!consecutive) {
    thrust::sort(policy, indices_data, indices_data + num_inp,
      [=] __device__ (int64_t a, int64_t b) -> bool {
        for (int64_t i = 0; i < n; ++i) {
          scalar_t lhs = c10::load(&input_flat_ptr[i + a * n]);
          scalar_t rhs = c10::load(&input_flat_ptr[i + b * n]);
          if (lhs < rhs) {
            return true;
          } else if (lhs > rhs) {
            return false;
          }
        }
        return false;
      }
    );
  }

  auto [inverse_indices, counts, num_out] = compute_unique(
    policy, indices_data, num_inp, indices,
    return_inverse, return_counts, options,
    [=] __device__ (int64_t a, int64_t b) -> bool {
      for (int64_t i = 0; i < n; ++i) {
        scalar_t lhs = c10::load(&input_flat_ptr[i + a * n]);
        scalar_t rhs = c10::load(&input_flat_ptr[i + b * n]);
        if (lhs != rhs) {
          return false;
        }
      }
      return true;
    },
    [=] __device__ (int64_t a, int64_t b) -> int64_t {
      for (int64_t i = 0; i < n; ++i) {
        scalar_t lhs = c10::load(&input_flat_ptr[i + a * n]);
        scalar_t rhs = c10::load(&input_flat_ptr[i + b * n]);
        if (lhs != rhs) {
          return 1;
        }
      }
      return 0;
    }
  );
  indices.resize_(num_out);

  return std::tuple<Tensor, Tensor, Tensor>(self.index_select(dim, indices), inverse_indices, counts);
}

} // namespace


std::tuple<Tensor, Tensor>
_unique_cuda(const Tensor& self, const bool sorted, const bool return_inverse) {
  return AT_DISPATCH_V2(self.scalar_type(), "unique", AT_WRAP([&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    auto [output, inverse, _] = internal::unique_cuda_template<scalar_t>(self, false, return_inverse, false);
    return std::make_tuple(output, inverse);
  }), AT_EXPAND(AT_ALL_TYPES), kBool, kBFloat16, kHalf, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
}

std::tuple<Tensor, Tensor, Tensor>
_unique2_cuda(const Tensor& self, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_V2(self.scalar_type(), "unique", AT_WRAP([&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    return internal::unique_cuda_template<scalar_t>(self, false, return_inverse, return_counts);
  }), AT_EXPAND(AT_ALL_TYPES), kBool, kBFloat16, kHalf, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
}

std::tuple<Tensor, Tensor, Tensor>
unique_dim_cuda(const Tensor& self, const int64_t dim, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_V2(self.scalar_type(), "unique_dim", AT_WRAP([&] {
    return unique_dim_cuda_template<scalar_t>(self, dim, false, return_inverse, return_counts);
  }), AT_EXPAND(AT_ALL_TYPES), kBool, kBFloat16, kHalf, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
}

std::tuple<Tensor, Tensor, Tensor>
unique_dim_consecutive_cuda(const Tensor& self, const int64_t dim, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_V2(self.scalar_type(), "unique_dim", AT_WRAP([&] {
    return unique_dim_cuda_template<scalar_t>(self, dim, true, return_inverse, return_counts);
  }), AT_EXPAND(AT_ALL_TYPES), kBool, kBFloat16, kHalf, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
}

std::tuple<Tensor, Tensor, Tensor>
unique_consecutive_cuda(const Tensor& self, const bool return_inverse, const bool return_counts, std::optional<int64_t> dim) {
  if (!dim.has_value()) {
    return AT_DISPATCH_V2(self.scalar_type(), "unique", AT_WRAP([&] {
      // The current CUDA implementation of unique always sort due to the
      // lack of hashtable implementation in thrust
      return internal::unique_cuda_template<scalar_t>(self, true, return_inverse, return_counts);
    }), AT_EXPAND(AT_ALL_TYPES), kBool, kBFloat16, kHalf, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
  }
  return unique_dim_consecutive_cuda(self, dim.value(), return_inverse, return_counts);
}

}  // namespace at::native
