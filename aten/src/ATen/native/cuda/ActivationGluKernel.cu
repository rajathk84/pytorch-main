#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>

namespace at::native {

// -----------------------------------
// glu forward
// -----------------------------------
void glu_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, iter.dtype(), "glu_cuda", [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a_, scalar_t b_) -> scalar_t {
          const opmath_t a = a_;
          const opmath_t b = b_;
          const opmath_t one = opmath_t(1);
          const opmath_t sigmoid = one / (one + std::exp(-b));
          return a * sigmoid;
        });
      });
}

// -----------------------------------
// glu forward ad
// -----------------------------------
void glu_jvp_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, iter.dtype(), "glu_cuda", [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(
                scalar_t res_, scalar_t b_, scalar_t da_, scalar_t db_)
                -> scalar_t {
              const opmath_t res = res_;
              const opmath_t b = b_;
              const opmath_t da = da_;
              const opmath_t db = db_;
              const opmath_t one = opmath_t(1);

              const opmath_t sig_b = one / (one + std::exp(-b));
              return (da * sig_b + res * (db - sig_b * db));
            });
      });
}

// -----------------------------------
// glu backward
// -----------------------------------

// Byte offsets don't require multiplication by sizeof(T), so are slightly
// cheaper. For fixed offsets, this removes all penalty from 64-bit indexing.
template <typename T>
__device__ T* byte_offset(T* ptr, int64_t offset) {
  using byte_ptr_t = typename std::
      conditional<std::is_const<T>::value, const char*, char*>::type;
  return reinterpret_cast<T*>(reinterpret_cast<byte_ptr_t>(ptr) + offset);
}

template <typename scalar_t, typename OffsetCalc>
__global__ void glu_backward_kernel(
    int numel,
    scalar_t* gI,
    const scalar_t* I,
    const scalar_t* gO,
    OffsetCalc offset_calculator,
    int64_t gI_byte_offset,
    int64_t I_byte_offset) {
  using opmath_t = at::opmath_type<scalar_t>;

  const uint32_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (linear_index >= numel) {
    return;
  }
  const auto offsets = offset_calculator.get(linear_index);

  // We explicitly iterate over the first half of the input tensor, and
  // gI_byte_offset and I_byte_offset are the offsets to access the
  // corresponding index in the second half of the tensor.
  const opmath_t a = I[offsets[1]];
  const opmath_t b = *byte_offset(I + offsets[1], I_byte_offset);
  const opmath_t gO_val = gO[offsets[2]];

  const auto one = opmath_t(1);
  const opmath_t sigmoid = one / (one + std::exp(-b));

  auto* gA = gI + offsets[0];
  *gA = sigmoid * gO_val;

  auto* gB = byte_offset(gA, gI_byte_offset);
  *gB = (one - sigmoid) * sigmoid * gO_val * a;
}

void launch_glu_backward_kernel(
    const TensorIteratorBase& iter,
    int64_t gI_stride,
    int64_t I_stride) {
  const auto N = iter.numel();
  TORCH_INTERNAL_ASSERT_DEBUG_ONLY(
      N > 0 && N <= std::numeric_limits<int32_t>::max());
  const auto offset_calculator = make_element_offset_calculator<3>(iter);
  constexpr int64_t block_size = 256;
  const int64_t grid = (N + block_size - 1) / block_size;
  const auto stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      kHalf, kBFloat16, iter.common_dtype(), "glu_backward_cuda", [&] {
        auto gI = static_cast<scalar_t*>(iter.data_ptr(0));
        auto I = static_cast<const scalar_t*>(iter.data_ptr(1));
        auto gO = static_cast<const scalar_t*>(iter.data_ptr(2));
        glu_backward_kernel<<<grid, block_size, 0, stream>>>(
            N,
            gI,
            I,
            gO,
            offset_calculator,
            gI_stride * sizeof(scalar_t),
            I_stride * sizeof(scalar_t));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

REGISTER_DISPATCH(glu_stub, &glu_kernel);
REGISTER_DISPATCH(glu_jvp_stub, &glu_jvp_kernel);

} // namespace at::native
