#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at::native {

template<typename scalar_t>
struct AbsFunctor {
  __device__ __forceinline__ scalar_t operator() (const scalar_t a) const {
    return std::abs(a);
  }
};

CONSTEXPR_EXCEPT_WIN_CUDA char abs_name[] = "abs_kernel";
void abs_kernel_cuda(TensorIteratorBase& iter) {
  auto dtype = iter.dtype();
  if (at::isComplexType(dtype)) {
#if AT_USE_JITERATOR()
    static const auto abs_string = jiterator_stringify(
        template <typename T> T abs_kernel(T x) { return std::abs(x); });
    AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "abs_cuda", [&]() {
      jitted_gpu_kernel<
          /*name=*/abs_name,
          /*return_dtype=*/scalar_t,
          /*common_dtype=*/scalar_t,
          /*arity=*/1>(iter, abs_string);
    });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "abs_cuda", [&]() {
      using opmath_t = at::opmath_type<scalar_t>;
      gpu_kernel(iter, AbsFunctor<opmath_t>());
    });
#endif
  } else {
    AT_DISPATCH_ALL_TYPES_AND3(
        ScalarType::Half,
        ScalarType::BFloat16,
        ScalarType::Bool,
        iter.dtype(),
        "abs_cuda",
        [&]() { gpu_kernel(iter, AbsFunctor<scalar_t>()); });
  }
}

  REGISTER_DISPATCH(abs_stub, &abs_kernel_cuda);

} // namespace at::native
