#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/UpSample.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/upsample_bicubic2d_native.h>
#include <ATen/ops/upsample_bicubic2d_backward_native.h>
#endif

namespace at::native {
namespace {

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bicubic2d_out_frame(
    const int num_elements,
    const accscalar_t height_scale,
    const accscalar_t width_scale,
    const bool align_corners,
    const PackedTensorAccessor64<const scalar_t, 4> idata,
    PackedTensorAccessor64<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int input_height = idata.size(2);
  const int input_width = idata.size(3);
  const int output_height = odata.size(2);
  const int output_width = odata.size(3);

  if (index >= num_elements) {
    return;
  }

  // Special case: input and output are the same size, just copy
  const int output_x = index % output_width;
  const int output_y = index / output_width;

  if (input_height == output_height && input_width == output_width) {
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; c++) {
        const scalar_t val = idata[n][c][output_y][output_x];
        odata[n][c][output_y][output_x] = val;
      }
    }
    return;
  }

  // Interpolation kernel
  accscalar_t real_x = area_pixel_compute_source_index(
      width_scale, output_x, align_corners, /*cubic=*/true);
  int in_x = floorf(real_x);
  accscalar_t t_x = real_x - in_x;

  accscalar_t real_y = area_pixel_compute_source_index(
      height_scale, output_y, align_corners, /*cubic=*/true);
  int in_y = floorf(real_y);
  accscalar_t t_y = real_y - in_y;

  for (int n = 0; n < batchsize; n++) {
    for (int c = 0; c < channels; c++) {
      accscalar_t coefficients[4];

      for (int k = 0; k < 4; k++) {
        coefficients[k] = cubic_interp1d(
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x - 1),
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x + 0),
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x + 1),
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x + 2),
            t_x);
      }

      odata[n][c][output_y][output_x] = static_cast<scalar_t>(cubic_interp1d(
          coefficients[0],
          coefficients[1],
          coefficients[2],
          coefficients[3],
          t_y));
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bicubic2d_backward_out_frame(
    const int num_elements,
    const accscalar_t height_scale,
    const accscalar_t width_scale,
    const bool align_corners,
    PackedTensorAccessor64<scalar_t, 4> idata,
    const PackedTensorAccessor64<const scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int input_height = idata.size(2);
  const int input_width = idata.size(3);
  const int output_height = odata.size(2);
  const int output_width = odata.size(3);

  if (index >= num_elements) {
    return;
  }

  const int output_x = index % output_width;
  const int output_y = index / output_width;
  // special case: output_xust copy
  if (input_height == output_height && input_width == output_width) {
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const scalar_t val = odata[n][c][output_y][output_x];
        idata[n][c][output_y][output_x] = val;
      }
    }
    return;
  }

  accscalar_t real_x = area_pixel_compute_source_index(
      width_scale, output_x, align_corners, /*cubic=*/true);
  int input_x = floorf(real_x);
  accscalar_t t_x = real_x - input_x;

  accscalar_t real_y = area_pixel_compute_source_index(
      height_scale, output_y, align_corners, /*cubic=*/true);
  int input_y = floorf(real_y);
  accscalar_t t_y = real_y - input_y;

  accscalar_t x_coeffs[4];
  accscalar_t y_coeffs[4];

  get_cubic_upsampling_coefficients(x_coeffs, t_x);
  get_cubic_upsampling_coefficients(y_coeffs, t_y);

  for (int n = 0; n < batchsize; n++) {
    for (int c = 0; c < channels; ++c) {
      scalar_t out_value = odata[n][c][output_y][output_x];
      for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
          upsample_increment_value_bounded<scalar_t, accscalar_t>(
              idata,
              n,
              c,
              input_height,
              input_width,
              input_y - 1 + i,
              input_x - 1 + j,
              out_value * y_coeffs[i] * x_coeffs[j]);
        }
      }
    }
  }
}

static void upsample_bicubic2d_out_cuda_template(
    const Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    std::optional<double> scales_h,
    std::optional<double> scales_w) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU(__func__, {input_arg, output_arg});

  int output_height = output_size[0];
  int output_width = output_size[1];

  int input_height = input.size(2);
  int input_width = input.size(3);

  output.zero_();

  const int num_output_elements = output_height * output_width;
  const int max_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);

  // Launch kernel
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      input.scalar_type(), "upsample_bicubic2d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<const scalar_t, 4>();
        auto odata = output.packed_accessor64<scalar_t, 4>();

        // Get scaling factors
        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_bicubic2d_out_frame<scalar_t, accscalar_t>
            <<<ceil_div(num_output_elements, max_threads),
               max_threads,
               0,
               stream>>>(
                num_output_elements,
                rheight,
                rwidth,
                align_corners,
                idata,
                odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_bicubic2d_backward_out_cuda_template(
    const Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    std::optional<double> scales_h,
    std::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(__func__, {grad_output_arg, grad_input_arg});

  int output_height = output_size[0];
  int output_width = output_size[1];

  int input_height = input_size[2];
  int input_width = input_size[3];

  Tensor grad_output = grad_output_.contiguous();

  grad_input.zero_();

  const int num_kernels = output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      grad_output.scalar_type(), "upsample_bicubic2d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.packed_accessor64<scalar_t, 4>();
        auto odata = grad_output.packed_accessor64<const scalar_t, 4>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_bicubic2d_backward_out_frame<scalar_t, accscalar_t>
            <<<ceil_div(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(
                num_kernels, rheight, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

TORCH_IMPL_FUNC(upsample_bicubic2d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& output) {
  upsample_bicubic2d_out_cuda_template(output, input, output_size, align_corners, scales_h, scales_w);
}

TORCH_IMPL_FUNC(upsample_bicubic2d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    std::optional<double> scales_h,
    std::optional<double> scales_w,
    const Tensor& grad_input) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bicubic2d_backward_out_cuda");
  upsample_bicubic2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_h, scales_w);
}

} // namespace at::native
