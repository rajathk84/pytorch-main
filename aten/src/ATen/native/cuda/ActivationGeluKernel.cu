#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>

namespace at::native {

void GeluCUDAKernelImpl(TensorIteratorBase& it, GeluType approximate) {
  if (approximate == GeluType::Tanh) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, it.dtype(), "GeluCUDAKernelImpl", [&]() {
      gpu_kernel(it, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
        using opmath_t = at::opmath_type<scalar_t>;
        constexpr opmath_t kBeta = M_SQRT2 * M_2_SQRTPI * opmath_t(0.5);
        constexpr opmath_t kKappa = 0.044715;
        auto x_cube = static_cast<opmath_t>(x) * static_cast<opmath_t>(x) * static_cast<opmath_t>(x);
        auto inner = kBeta * (static_cast<opmath_t>(x) + kKappa * x_cube);
        return opmath_t(0.5) * static_cast<opmath_t>(x) * (opmath_t(1) + c10::cuda::compat::tanh(inner));
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, it.dtype(), "GeluCUDAKernelImpl", [&]() {
      gpu_kernel(it, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
        using opmath_t = at::opmath_type<scalar_t>;
        constexpr opmath_t kAlpha = M_SQRT1_2;
        return static_cast<opmath_t>(x) * opmath_t(0.5) * (opmath_t(1) + ::erf(static_cast<opmath_t>(x) * kAlpha));
      });
    });
  }
}

void GeluBackwardCUDAKernelImpl(TensorIteratorBase& it, GeluType approximate) {
  if (approximate == GeluType::Tanh) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
        it.dtype(), "GeluBackwardCUDAKernelImpl", [&]() {
          gpu_kernel(it, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            constexpr opmath_t kBeta = M_SQRT2 * M_2_SQRTPI * opmath_t(0.5);
            constexpr opmath_t kKappa = 0.044715;
            auto x_sq = static_cast<opmath_t>(x) * static_cast<opmath_t>(x);
            auto x_cube = x_sq * static_cast<opmath_t>(x);
            auto inner = kBeta * (static_cast<opmath_t>(x) + kKappa * x_cube);
            auto tanh_inner = c10::cuda::compat::tanh(inner);

            auto left = opmath_t(0.5) * static_cast<opmath_t>(x);
            auto right = opmath_t(1) + tanh_inner;

            auto left_derivative = opmath_t(0.5) * right;

            auto tanh_derivative = opmath_t(1) - tanh_inner * tanh_inner;
            auto inner_derivative = kBeta * (opmath_t(1) + opmath_t(3) * kKappa * x_sq);
            auto right_derivative = left * tanh_derivative * inner_derivative;

            return static_cast<opmath_t>(dy) * (left_derivative + right_derivative);
        });
      });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
        it.dtype(), "GeluBackwardCUDAKernelImpl", [&]() {
          gpu_kernel(it, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            constexpr opmath_t kBeta = M_2_SQRTPI * M_SQRT1_2 * opmath_t(0.5);
            constexpr opmath_t kAlpha = M_SQRT1_2;
            const opmath_t cdf =
                opmath_t(0.5) * (opmath_t(1) + ::erf(static_cast<opmath_t>(x) * kAlpha));
            const opmath_t pdf =
                c10::cuda::compat::exp(
                    opmath_t(-0.5) * static_cast<opmath_t>(x) * static_cast<opmath_t>(x)) *
                kBeta;
            return static_cast<opmath_t>(dy) * (cdf + static_cast<opmath_t>(x) * pdf);
          });
        });
  }
}

} // namespace at::native
