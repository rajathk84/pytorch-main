#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char cosh_name[] = "cosh_impl";
#endif

void cosh_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR()
    static const auto cosh_string = jiterator_stringify(
        template <typename T> T cosh_impl(T a) { return std::cosh(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "cosh_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/cosh_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, cosh_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "cosh_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::cosh(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "cosh_cuda",
        [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::cosh(a);
          });
        });
  }
}

REGISTER_DISPATCH(cosh_stub, &cosh_kernel_cuda);

} // namespace at::native
