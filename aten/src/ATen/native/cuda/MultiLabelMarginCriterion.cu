#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <c10/macros/Macros.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/block_reduce.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/CUDAFunctions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/zeros_like.h>
#include <ATen/ops/sum_cuda_dispatch.h>
#include <ATen/ops/multilabel_margin_loss.h>
#endif


namespace at::native {

namespace {
const int MULTILABELMARGIN_THREADS = 128;

void multilabel_margin_loss_shape_check(
    int64_t& nframe,
    int64_t& dim,
    const int64_t& ndims,
    const Tensor& input,
    const Tensor& target) {
    TORCH_CHECK(
        (ndims == 2 && input.size(1) != 0) || (ndims == 1 && input.size(0) != 0) || ndims == 0,
        "Expected non-empty vector or matrix with optional 0-dim batch size, but got: ",
        input.sizes());

    if (ndims <= 1) {
      nframe = 1;
      dim = ndims == 0 ? 1 : input.size(0);
      TORCH_CHECK(
          target.dim() <= 1 && target.numel() == dim,
          "inconsistent target size: ", target.sizes(), " for input of size: ",
          input.sizes());
    } else {
      nframe = input.size(0);
      dim = input.size(1);
      TORCH_CHECK(
          target.dim() == 2 && target.size(0) == nframe &&
          target.size(1) == dim,
          "inconsistent target size: ", target.sizes(), " for input of size: ",
          input.sizes());
    }
}

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(MULTILABELMARGIN_THREADS)
__global__ void multilabel_margin_loss_forward_kernel(
    scalar_t* output,
    const scalar_t* input,
    const int64_t* target,
    scalar_t* is_target,
    int nframe,
    int dim,
    bool size_average) {

  // vectors:
  int k = blockIdx.x;
  const scalar_t* input_k = input + k * dim;
  const int64_t* target_k = target + k * dim;
  scalar_t* output_k = output + k;
  scalar_t* is_target_k = is_target + k * dim;

  // zero is_target
  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    is_target_k[d] = static_cast<scalar_t>(0);
  }
  __syncthreads();

  // mark targets in is_target
  if (threadIdx.x == 0) {
    for (int dt = 0; dt < dim; dt++) {
      int target_idx = target_k[dt];
      if (target_idx < 0) {
        break;
      }
      is_target_k[target_idx] = static_cast<scalar_t>(1);
    }
  }
  __syncthreads();

  // iterate over targets
  accscalar_t sum = 0;
  for (int dt = 0; dt < dim; dt++) {
    // next target:
    int target_idx = target_k[dt];
    if (target_idx < 0) {
      break;
    }

    // current value for target
    scalar_t input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    for (int d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!static_cast<int>(is_target_k[d])) {
        scalar_t z = 1 - input_target_k + input_k[d];
        if (z > 0) {
          sum += z;
        }
      }
    }
  }

  // Temporary sums (for mapreduce)
  __shared__ accscalar_t smem[MULTILABELMARGIN_THREADS];
  accscalar_t total_sum = cuda_utils::BlockReduceSum(sum, smem);
  if (threadIdx.x == 0) {
    if (size_average) {
      *output_k = static_cast<scalar_t>((total_sum / dim) / nframe);
    } else {
      *output_k = static_cast<scalar_t>(total_sum / dim);
    }
  }
}

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(MULTILABELMARGIN_THREADS)
__global__ void multilabel_margin_loss_backward_kernel(
    scalar_t* grad_input,
    const scalar_t* grad_output,
    const scalar_t* input,
    const int64_t* target,
    const scalar_t* is_target,
    int nframe,
    int dim,
    bool size_average,
    bool reduce) {

  int k = blockIdx.x;
  const scalar_t* input_k = input + k * dim;
  scalar_t* grad_input_k = grad_input + k * dim;
  const int64_t* target_k = target + k * dim;
  const scalar_t* is_target_k = is_target + k * dim;

  const scalar_t* grad_output_k = grad_output;
  if (!reduce) {
    grad_output_k += k;
  }

  // gain:
  scalar_t g = static_cast<scalar_t>(
      size_average && reduce ? 1. / static_cast<accscalar_t>(nframe * dim)
                             : 1. / static_cast<accscalar_t>(dim));

  // zero gradients:
  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    grad_input_k[d] = static_cast<scalar_t>(0);
  }
  __syncthreads();

  // iterate over targets
  for (int dt = 0; dt < dim; dt++) {
    // next target:
    int target_idx = static_cast<int>(target_k[dt]);
    if (target_idx < 0) {
      break;
    }

    // current value for target
    scalar_t input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    accscalar_t sum = 0;
    for (int d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!static_cast<int>(is_target_k[d])) {
        scalar_t z = 1 - input_target_k + input_k[d];
        if (z > 0) {
          sum -= g;
          grad_input_k[d] += g;
        }
      }
    }
    __syncthreads();

    // Temporary sums (for mapreduce)
    __shared__ accscalar_t smem[MULTILABELMARGIN_THREADS];
    accscalar_t total_sum = cuda_utils::BlockReduceSum(sum, smem);
    if (threadIdx.x == 0) {
      grad_input_k[target_idx] += static_cast<scalar_t>(total_sum);
    }
  }

  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    grad_input_k[d] *= *grad_output_k;
  }
}

void multilabel_margin_loss_forward_out_cuda_template(
    const Tensor& input,
    const Tensor& target,
    int64_t reduction,
    Tensor& output,
    Tensor& is_target) {
  int64_t nframe, dim;
  const int64_t ndims = input.dim();
  multilabel_margin_loss_shape_check(nframe, dim, ndims, input, target);

  if (input.numel() == 0) {
    return;
  }

  auto input_ = input.contiguous();
  auto target_ = target.contiguous();
  auto is_target_ = is_target.contiguous();
  is_target_.resize_as_(target);

  if (input.dim() <= 1) {
    output.resize_({});

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "multilabel_margin_loss_forward_kernel",
        [&] {
          using accscalar_t = at::acc_type<scalar_t, true>;
          multilabel_margin_loss_forward_kernel<scalar_t, accscalar_t>
              <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                  output.mutable_data_ptr<scalar_t>(),
                  input_.const_data_ptr<scalar_t>(),
                  target_.const_data_ptr<int64_t>(),
                  is_target_.mutable_data_ptr<scalar_t>(),
                  1,
                  dim,
                  reduction == at::Reduction::Mean);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  } else if (input.dim() == 2) {
    dim3 blocks(input.size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    if (reduction != at::Reduction::None) {
      auto output_tmp = at::empty({input_.size(0)}, input_.options());
      output.resize_({});
      AT_DISPATCH_FLOATING_TYPES_AND2(
          at::ScalarType::Half,
          at::ScalarType::BFloat16,
          input.scalar_type(),
          "multilabel_margin_loss_forward_kernel",
          [&] {
            using accscalar_t = at::acc_type<scalar_t, true>;
            multilabel_margin_loss_forward_kernel<scalar_t, accscalar_t>
                <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                    output_tmp.mutable_data_ptr<scalar_t>(),
                    input_.const_data_ptr<scalar_t>(),
                    target_.const_data_ptr<int64_t>(),
                    is_target_.mutable_data_ptr<scalar_t>(),
                    nframe,
                    dim,
                    reduction == at::Reduction::Mean);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
      at::cuda::sum_out(
          output,
          output_tmp,
          at::IntArrayRef(std::vector<int64_t>{}),
          false,
          output.scalar_type());
    } else {
      output.resize_({input.size(0)});
      AT_DISPATCH_FLOATING_TYPES_AND2(
          at::ScalarType::Half,
          at::ScalarType::BFloat16,
          input.scalar_type(),
          "multilabel_margin_loss_forward_kernel",
          [&] {
            using accscalar_t = at::acc_type<scalar_t, true>;
            multilabel_margin_loss_forward_kernel<scalar_t, accscalar_t>
                <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                    output.mutable_data_ptr<scalar_t>(),
                    input_.const_data_ptr<scalar_t>(),
                    target_.const_data_ptr<int64_t>(),
                    is_target_.mutable_data_ptr<scalar_t>(),
                    nframe,
                    dim,
                    false);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
    }

  } else {
    TORCH_CHECK(
        false,
        "Expected 2D input with optional zero batch dim, or 1D input with non-zero dims, but got sizes: ",
        input.sizes());
  }
}

void multilabel_margin_loss_backward_cuda_out_template(
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& target,
    int64_t reduction,
    const Tensor& is_target,
    Tensor& grad_input) {
  int64_t nframe, dim;
  const int64_t ndims = input.dim();
  multilabel_margin_loss_shape_check(nframe, dim, ndims, input, target);

  if (input.numel() == 0) {
    return;
  }

  auto input_ = input.contiguous();
  auto target_ = target.contiguous();
  auto is_target_ = is_target.contiguous();
  auto grad_output_ = grad_output.contiguous();
  grad_input.resize_as_(input_);

  if (grad_input.dim() <= 1) {
    int target_size = target_.dim() == 0 ? 1 : target_.size(0);
    TORCH_CHECK(
        (target_.numel() != 0) && (target_.dim() <= 1) && (target_size == dim),
        "inconsistent target size");
    TORCH_CHECK(
        target_.sizes() == is_target_.sizes(), "inconsistent is_target size");
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "multilabel_margin_loss_backward_kernel",
        [&] {
          using accscalar_t = at::acc_type<scalar_t, true>;
          multilabel_margin_loss_backward_kernel<scalar_t, accscalar_t>
              <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
                  grad_input.mutable_data_ptr<scalar_t>(),
                  grad_output_.const_data_ptr<scalar_t>(),
                  input_.const_data_ptr<scalar_t>(),
                  target_.const_data_ptr<int64_t>(),
                  is_target_.const_data_ptr<scalar_t>(),
                  1,
                  dim,
                  reduction == at::Reduction::Mean,
                  reduction != at::Reduction::None);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  } else if (grad_input.dim() == 2) {
    TORCH_CHECK(
        (input_.size(1) != 0) && (target_.dim() == 2) &&
            (target_.size(0) == nframe) && (target_.size(1) == dim),
        "inconsistent target size");
    TORCH_CHECK(target_.sizes() == is_target_.sizes(), "inconsistent is_target size");
    dim3 blocks(grad_input.size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        input.scalar_type(),
        "multilabel_margin_loss_backward_kernel",
        [&] {
          using accscalar_t = at::acc_type<scalar_t, true>;
          multilabel_margin_loss_backward_kernel<scalar_t, accscalar_t>
              <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
                  grad_input.mutable_data_ptr<scalar_t>(),
                  grad_output_.const_data_ptr<scalar_t>(),
                  input_.const_data_ptr<scalar_t>(),
                  target_.const_data_ptr<int64_t>(),
                  is_target_.const_data_ptr<scalar_t>(),
                  grad_input.size(0),
                  grad_input.size(1),
                  reduction == at::Reduction::Mean,
                  reduction != at::Reduction::None);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  } else {
    TORCH_CHECK(
        false,
        "Expected 2D input with optional zero batch dim, or 1D input with non-zero dims, but got sizes: ",
        grad_input.sizes());
  }
}

} // namespace

std::tuple<Tensor&, Tensor&> multilabel_margin_loss_forward_out_cuda(
    const Tensor& self,
    const Tensor& target,
    int64_t reduction,
    Tensor& output,
    Tensor& is_target) {
  multilabel_margin_loss_forward_out_cuda_template(
      self, target, reduction, output, is_target);
  return std::tuple<Tensor&, Tensor&>(output, is_target);
}

std::tuple<Tensor, Tensor> multilabel_margin_loss_forward_cuda(
    const Tensor& self,
    const Tensor& target,
    int64_t reduction) {
  auto output = at::empty({0}, self.options());
  auto is_target = at::empty({0}, self.options());
  multilabel_margin_loss_forward_out_cuda_template(
      self, target, reduction, output, is_target);
  return std::make_tuple(output, is_target);
}

Tensor& multilabel_margin_loss_backward_cuda_out(
    const Tensor& grad_output,
    const Tensor& self,
    const Tensor& target,
    int64_t reduction,
    const Tensor& is_target,
    Tensor& grad_input) {
  multilabel_margin_loss_backward_cuda_out_template(
      grad_output, self, target, reduction, is_target, grad_input);
  return grad_input;
}

Tensor multilabel_margin_loss_backward_cuda(
    const Tensor& grad_output,
    const Tensor& self,
    const Tensor& target,
    int64_t reduction,
    const Tensor& is_target) {
  auto grad_input = at::zeros_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  multilabel_margin_loss_backward_cuda_out_template(
      grad_output, self, target, reduction, is_target, grad_input);
  return grad_input;
}

} // namespace at::native
