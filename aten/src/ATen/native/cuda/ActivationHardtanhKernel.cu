#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>

namespace at::native {
namespace {

void hardtanh_backward_kernel(
    TensorIterator& iter,
    const Scalar& min,
    const Scalar& max) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      iter.dtype(), "hardtanh_backward_cuda", [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        auto min_val = min.to<opmath_t>();
        auto max_val = max.to<opmath_t>();
        gpu_kernel(
            iter,
            [min_val, max_val] GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
              opmath_t aop = static_cast<opmath_t>(a);
              opmath_t bop = static_cast<opmath_t>(b);
              return (bop <= min_val) || (bop >= max_val) ? opmath_t(0) : aop;
            });
      });
}
} // namespace

REGISTER_DISPATCH(hardtanh_backward_stub, &hardtanh_backward_kernel);

} // namespace at::native
