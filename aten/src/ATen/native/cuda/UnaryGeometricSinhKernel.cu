#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char sinh_name[] = "sinh_impl";
#endif

void sinh_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR()
    static const auto sinh_string = jiterator_stringify(
        template <typename T> T sinh_impl(T a) { return std::sinh(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "sinh_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/sinh_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, sinh_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "sinh_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::sinh(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "sinh_cuda",
        [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::sinh(a);
          });
        });
  }
}

REGISTER_DISPATCH(sinh_stub, &sinh_kernel_cuda);

} // namespace at::native
