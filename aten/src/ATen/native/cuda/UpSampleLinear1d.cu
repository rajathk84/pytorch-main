#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/UpSample.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/upsample_linear1d_native.h>
#include <ATen/ops/upsample_linear1d_backward_native.h>
#endif

namespace at::native {
namespace {

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(512)
__global__ void upsample_linear1d_out_frame(
    const int n,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor64<const scalar_t, 3> idata,
    PackedTensorAccessor64<scalar_t, 3> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int width1 = idata.size(2);
  const int width2 = odata.size(2);

  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][w1];
          odata[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val =
            w0lambda * idata[n][c][w1] + w1lambda * idata[n][c][w1 + w1p];
        odata[n][c][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(512)
__global__ void upsample_linear1d_out_frame_backward(
    const int n,
    const accscalar_t rwidth,
    const bool align_corners,
    PackedTensorAccessor64<scalar_t, 3> idata,
    const PackedTensorAccessor64<const scalar_t, 3> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int width1 = idata.size(2);
  const int width2 = odata.size(2);

  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = odata[n][c][w1];
          idata[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const scalar_t d2val = odata[n][c][w2];
        gpuAtomicAddNoReturn(&idata[n][c][w1], static_cast<scalar_t>(w0lambda * d2val));
        gpuAtomicAddNoReturn(
            &idata[n][c][w1 + w1p], static_cast<scalar_t>(w1lambda * d2val));
      }
    }
  }
}

static void upsample_linear1d_out_cuda_template(
    const Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    std::optional<double> scales) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU(__func__, {input_arg, output_arg});

  int output_width = output_size[0];

  int input_width = input.size(2);

  output.zero_();

  AT_ASSERT(input_width > 0 && output_width > 0);

  const int num_kernels = output_width;
  const int num_threads = 512;
      //at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      input.scalar_type(), "upsample_linear1d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<const scalar_t, 3>();
        auto odata = output.packed_accessor64<scalar_t, 3>();

        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
          input_width, output_width, align_corners, scales);

        upsample_linear1d_out_frame<scalar_t, accscalar_t>
            <<<ceil_div(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(num_kernels, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_linear1d_backward_out_cuda_template(
    const Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    std::optional<double> scales) {
  TensorArg grad_output_arg{grad_output_, "grad_output_", 1},
      grad_input_arg{grad_input, "grad_input", 2};
  checkAllSameGPU(__func__, {grad_output_arg, grad_input_arg});

  int output_width = output_size[0];

  int input_width = input_size[2];

  Tensor grad_output = grad_output_.contiguous();

  grad_input.zero_();

  const int num_kernels = output_width;
  const int num_threads = 512;
      //at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      grad_output.scalar_type(), "upsample_linear1d_out_frame_backward", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.packed_accessor64<scalar_t, 3>();
        auto odata = grad_output.packed_accessor64<const scalar_t, 3>();

        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales);

        upsample_linear1d_out_frame_backward<scalar_t, accscalar_t>
            <<<ceil_div(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(num_kernels, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

TORCH_IMPL_FUNC(upsample_linear1d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    std::optional<double> scales,
    const Tensor& output
) {
  upsample_linear1d_out_cuda_template(output, input, output_size, align_corners, scales);
}

TORCH_IMPL_FUNC(upsample_linear1d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    std::optional<double> scales,
    const Tensor& grad_input
) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_linear1d_backward_out_cuda");
  upsample_linear1d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales);
}

} // namespace at::native
