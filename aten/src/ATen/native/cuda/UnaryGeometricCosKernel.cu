#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char cos_name[] = "cos_impl";
#endif // AT_USE_JITERATOR()

void cos_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR()
    static const auto cos_string = jiterator_stringify(
        template <typename T> T cos_impl(T a) { return std::cos(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "cos_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/cos_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, cos_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "cos_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::cos(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "cos_cuda",
        [&]() {
          gpu_kernel(
              iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t { return ::cos(a); });
        });
  }
}

REGISTER_DISPATCH(cos_stub, &cos_kernel_cuda);

} // namespace at::native
