#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/cuda/Distributions.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

/**
 * Note [Register spilling in hiprand call for CUDA < 10]
 * ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 * For CUDA < 10, hiprandStatePhilox4_32_10_t engine achieves poor performance (60% SOL bandwidth)
 * when called to generate one random number at a time. This is because the line
 *            unsigned ret = (&state->output.x)[state->STATE++];
 * in
 *            QUALIFIERS unsigned int hiprand(hiprandStatePhilox4_32_10_t *state)
 * in hiprand/hiprand_kernel.h dynamically indexes into state.output, preventing the compiler from ever
 * storing state.output in registers.
 *
 * CUDA 10 fixed this problem. However, for backwards compatibility, in the following kernels
 * we are using hiprand distributions that utilize curand4 call. curand4 call doesn't have the
 * register spilling problem.
 */

namespace {

template <typename scalar_t>
void poisson_cuda_kernel(
    const at::TensorBase &ret,
    const at::TensorBase &lambda,
    at::PhiloxCudaState philox_args) {
  auto functor = [philox_args] __device__(
          scalar_t & ret_val, const scalar_t& lambda) {
        CUDA_KERNEL_ASSERT(lambda >= 0 && "invalid Poisson rate, expected rate to be non-negative");
        auto seeds = at::cuda::philox::unpack(philox_args);
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(std::get<0>(seeds),
                    blockIdx.x * blockDim.x + threadIdx.x,
                    std::get<1>(seeds),
                    &state);
        ret_val = static_cast<scalar_t>(hiprand_poisson(&state, lambda));
      };
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t, decltype(functor),
                               /*max_threads_per_block=*/512,
                               /*min_blocks_per_sm==*/2>(ret, lambda, functor);
}

struct curand_uniform_wrapper {
  hiprandStatePhilox4_32_10_t &state;
  __device__ curand_uniform_wrapper(hiprandStatePhilox4_32_10_t &state): state(state) {}
  __device__ float operator()() {

  uint32_t val = hiprand(&state); //need just bits
  constexpr auto MASK = static_cast<uint32_t>((static_cast<uint64_t>(1) << std::numeric_limits<float>::digits) - 1);
  constexpr auto DIVISOR = static_cast<float>(1) / (static_cast<uint32_t>(1) << std::numeric_limits<float>::digits);
    return (val & MASK) * DIVISOR;
  }
};

template <typename scalar_t>
void binomial_cuda_kernel(
    at::TensorIteratorBase &iter,
    at::PhiloxCudaState philox_args) {
  using accscalar_t = at::acc_type<scalar_t, true>;

  at::native::distribution_binary_kernel(iter, philox_args,
      [] GPU_LAMBDA (hiprandStatePhilox4_32_10_t& state, scalar_t count, scalar_t prob) {
        #if defined(__CUDA_ARCH__) || defined(USE_ROCM)
        auto uniform_lambda = curand_uniform_wrapper(state);
        BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);
        auto sample = sample_binomial<scalar_t, accscalar_t, decltype(uniform_lambda)>(count, prob, standard_uniform);
        return static_cast<scalar_t>(sample);
        #else
        return count; // useless.
        #endif
      }
  );
}

template <typename scalar_t>
void gamma_cuda_kernel(
    const at::TensorBase &ret,
    const at::TensorBase &alpha,
    at::PhiloxCudaState philox_args) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  auto functor = [philox_args] __device__(
          scalar_t & ret_val, const scalar_t& alpha) {
        auto seeds = at::cuda::philox::unpack(philox_args);
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(std::get<0>(seeds),
                    blockIdx.x * blockDim.x + threadIdx.x,
                    std::get<1>(seeds),
                    &state);

        auto uniform_lambda = [&state] __device__ () {
          return hiprand_uniform(&state);
        };
        BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);

        auto normal_lambda = [&state] __device__ () {
          return hiprand_normal(&state);
        };
        BaseSampler<accscalar_t, decltype(normal_lambda)> standard_normal(normal_lambda);
        auto sample = sample_gamma<scalar_t, accscalar_t, decltype(uniform_lambda), decltype(normal_lambda)>(alpha, standard_uniform, standard_normal);
        auto min_value = std::numeric_limits<scalar_t>::min();
        ret_val = (min_value > sample) ? min_value : sample;
      };
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t, decltype(functor),
                               /*max_threads_per_block=*/256,
                               /*min_blocks_per_sm==*/2>(ret, alpha, functor);
}

} // namespace

namespace at::native {

void launch_dirichlet_kernel(at::TensorIteratorBase &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
                                  iter.input_dtype(), "dirichlet_cuda", [&] {
    at::native::gpu_kernel(
        iter,
        [] GPU_LAMBDA (scalar_t gamma, scalar_t gamma_sum) {
      auto ret_val = gamma / gamma_sum;
      auto min_value = std::numeric_limits<scalar_t>::min();
      auto max_value = 1 - std::numeric_limits<scalar_t>::epsilon();
      ret_val = (min_value > ret_val) ? min_value : ret_val;
      ret_val = (max_value < ret_val) ? max_value : ret_val;
      return ret_val;
    });
  });
}

void launch_poisson_cuda_kernel(
    const TensorBase &ret, const TensorBase &lambda, CUDAGeneratorImpl *gen) {
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(20);
  }
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "poisson_cuda", [&] {
    poisson_cuda_kernel<scalar_t>(ret, lambda, rng_engine_inputs);
  });
}

void launch_binomial_cuda_kernel(
    TensorIteratorBase &iter, CUDAGeneratorImpl *gen) {
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(42);
  }
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "binomial_cuda", [&] {
    binomial_cuda_kernel<scalar_t>(iter, rng_engine_inputs);
  });
}

void launch_gamma_kernel(
    const TensorBase &ret, const TensorBase &alpha, CUDAGeneratorImpl *gen) {
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(10);
  }
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "gamma_cuda", [&] {
     gamma_cuda_kernel<scalar_t>(ret, alpha, rng_engine_inputs);
   });
}

void launch_standard_gamma_grad_kernel(TensorIteratorBase &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "_standard_gamma_grad_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    gpu_kernel(iter,
      [] GPU_LAMBDA (scalar_t self_val, scalar_t output_val) {
        return standard_gamma_grad_one<scalar_t, accscalar_t>(self_val, output_val);
      });
  });
}

void launch_dirichlet_grad_kernel(TensorIteratorBase &iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.input_dtype(), "_dirichlet_grad_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    at::native::gpu_kernel(iter,
      [] GPU_LAMBDA (scalar_t x_val, scalar_t alpha_val, scalar_t total_val) -> scalar_t {
        return dirichlet_grad_one<scalar_t, accscalar_t>(x_val, alpha_val, total_val);
      });
  });
}

} // namespace at::native
