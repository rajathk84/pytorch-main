#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/Cross.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>

namespace at::native {

template <typename T, typename OffsetCalc, typename StrideType>
__global__ void cross_kernel(
    int numel, T* out, const T* x1, const T* x2, OffsetCalc offset_calculator,
    StrideType ostride, StrideType x1stride, StrideType x2stride) {
  CUDA_KERNEL_LOOP(i, numel) {
    const auto offsets = offset_calculator.get(i);
    auto* out_row = out + offsets[0];
    const auto* x1_row = x1 + offsets[1];
    const auto* x2_row = x2 + offsets[2];

    const T val0 = (x1_row[1 * x1stride] * x2_row[2 * x2stride] -
                    x1_row[2 * x1stride] * x2_row[1 * x2stride]);

    const T val1 = (x1_row[2 * x1stride] * x2_row[0 * x2stride] -
                    x1_row[0 * x1stride] * x2_row[2 * x2stride]);

    const T val2 = (x1_row[0 * x1stride] * x2_row[1 * x2stride] -
                    x1_row[1 * x1stride] * x2_row[0 * x2stride]);


    out_row[0 * ostride] = val0;
    out_row[1 * ostride] = val1;
    out_row[2 * ostride] = val2;
  }
}

void launch_cross_kernel(const TensorIteratorBase& iter, int64_t ostride,
                         int64_t x1stride, int64_t x2stride) {
  const auto N = iter.numel();
  auto offset_calculator = make_element_offset_calculator<3>(iter);
  TORCH_INTERNAL_ASSERT_DEBUG_ONLY(N > 0 && N <= std::numeric_limits<int32_t>::max());
  int64_t grid = (N + num_threads() - 1) / num_threads();
  auto stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.common_dtype(), "cross_cuda", [&] {
    auto out = static_cast<scalar_t*>(iter.data_ptr(0));
    auto x1 = static_cast<const scalar_t*>(iter.data_ptr(1));
    auto x2 = static_cast<const scalar_t*>(iter.data_ptr(2));
    constexpr int64_t int_max = std::numeric_limits<int>::max();
    if (ostride * 2 > int_max || x1stride * 2 > int_max || x2stride * 2 > int_max) {
      cross_kernel<<<grid, num_threads(), 0, stream>>>(
          N, out, x1, x2, offset_calculator, ostride, x1stride, x2stride);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      cross_kernel<<<grid, num_threads(), 0, stream>>>(
          N, out, x1, x2, offset_calculator,
          static_cast<int>(ostride),
          static_cast<int>(x1stride),
          static_cast<int>(x2stride));
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  });
}

void cross_impl(const Tensor& result, const Tensor& x1, const Tensor& x2, int64_t dim) {
  const int64_t ostride = result.stride(dim);
  const int64_t x1stride = x1.stride(dim);
  const int64_t x2stride = x2.stride(dim);

  auto iter = TensorIteratorConfig()
      .add_output(result)
      .add_const_input(x1)
      .add_const_input(x2)
      .resize_outputs(false)
      .declare_static_shape(result.sizes(), /*squash_dims=*/dim)
      .build();

  if (iter.numel() == 0) {
    return;
  }

  if (iter.can_use_32bit_indexing()) {
    launch_cross_kernel(iter, ostride, x1stride, x2stride);
  } else {
    for (auto&& sub_iter: iter.with_32bit_indexing()) {
      launch_cross_kernel(sub_iter, ostride, x1stride, x2stride);
    }
  }
}

REGISTER_DISPATCH(cross_stub, &cross_impl);

} // namespace at::native
