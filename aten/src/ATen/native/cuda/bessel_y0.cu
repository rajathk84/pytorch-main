#define TORCH_ASSERT_NO_OPERATORS

#include <ATen/native/UnaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/Math.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/cuda/jit_utils.h>
#include <ATen/NumericUtils.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/util/complex.h>

namespace at::native {
        namespace {
            CONSTEXPR_EXCEPT_WIN_CUDA char bessel_y0_name[] = "bessel_y0_forward";

            void bessel_y0_kernel_cuda(TensorIteratorBase& iterator) {
#if AT_USE_JITERATOR()
                AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "bessel_y0_cuda", [&]() {
                    jitted_gpu_kernel<bessel_y0_name, scalar_t, scalar_t, 1>(iterator, bessel_y0_string);
                });
#else
                AT_DISPATCH_FLOATING_TYPES(iterator.common_dtype(), "bessel_y0_cuda", [&]() {
                    gpu_kernel(iterator, []GPU_LAMBDA(scalar_t a) -> scalar_t {
                        return bessel_y0_forward(a);
                    });
                });
#endif // AT_USE_JITERATOR()
            }
        }

        REGISTER_DISPATCH(special_bessel_y0_stub, &bessel_y0_kernel_cuda);
} // namespace at::native
