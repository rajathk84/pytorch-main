#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/div_rtn.h>

#include <ATen/cuda/HIPContext.h>

#include <ATen/native/cuda/im2col.cuh>
#include <ATen/native/im2col_shape_check.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/col2im_native.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/im2col_native.h>
#endif

namespace at::native {
namespace {

void col2im_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    IntArrayRef output_size,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  TensorArg input_arg{input_, "input", 1};
  TensorArg output_arg{output, "output", 2};
  checkAllSameGPU(__func__, {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  TORCH_CHECK(
      kernel_size.size() == 2,
      "It is expected kernel_size equals to 2, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 2,
      "It is expected dilation equals to 2, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 2,
      "It is expected padding equals to 2, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 2,
      "It is expected stride equals to 2, but got size ",
      stride.size());

  int64_t output_height = output_size[0];
  int64_t output_width = output_size[1];
  int64_t kernel_height = kernel_size[0];
  int64_t kernel_width = kernel_size[1];
  int64_t dilation_height = dilation[0];
  int64_t dilation_width = dilation[1];
  int64_t pad_height = padding[0];
  int64_t pad_width = padding[1];
  int64_t stride_height = stride[0];
  int64_t stride_width = stride[1];

  col2im_shape_check(
      input_,
      Tensor(),
      output_height,
      output_width,
      kernel_height,
      kernel_width,
      dilation_height,
      dilation_width,
      pad_height,
      pad_width,
      stride_height,
      stride_width);

  Tensor input = input_.contiguous();

  bool batched_input = true;
  if (input.dim() == 2) {
    // Force batch
    batched_input = false;
    input = input.unsqueeze(0);
  }

  int64_t batch_size = input.size(0);
  int64_t n_input_plane = input.size(1);
  int64_t n_output_plane = n_input_plane / (kernel_width * kernel_height);
  int64_t input_batch_stride = input.stride(0);

  output.resize_({batch_size, n_output_plane, output_height, output_width});
  int64_t output_batch_stride = output.stride(0);

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND3(kHalf, kBFloat16, kBool,
      input.scalar_type(), "col2im_out_cuda", [&] {
    int64_t height_col = (output_height + 2 * pad_height -
                          (dilation_height * (kernel_height - 1) + 1)) /
            stride_height +
        1;
    int64_t width_col = (output_width + 2 * pad_width -
                         (dilation_width * (kernel_width - 1) + 1)) /
            stride_width +
        1;

    col2im_batched(
        at::cuda::getCurrentCUDAStream(),
        input.const_data_ptr<scalar_t>(),
        input_batch_stride,
        batch_size,
        n_output_plane,
        output_height,
        output_width,
        height_col,
        width_col,
        kernel_height,
        kernel_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        dilation_height,
        dilation_width,
        output.mutable_data_ptr<scalar_t>(),
        output_batch_stride);

  });
  if (!batched_input) {
    output = output.squeeze(0);
  }
}

} // namespace

Tensor& col2im_out_cuda(const Tensor& input,
    IntArrayRef output_size,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride,
    Tensor& output) {
  col2im_out_cuda_template(
      output, input, output_size, kernel_size, dilation, padding, stride);
  return output;
}

Tensor col2im_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  col2im_out_cuda_template(
      output, input, output_size, kernel_size, dilation, padding, stride);
  return output;
}

} // namespace at::native
