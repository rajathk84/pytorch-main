#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char sin_name[] = "sin_impl";
#endif

void sin_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR()
    static const auto sin_string = jiterator_stringify(
        template <typename T> T sin_impl(T a) { return std::sin(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "sin_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/sin_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, sin_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "sin_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::sin(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "sin_cuda",
        [&]() {
          gpu_kernel(
              iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t { return ::sin(a); });
        });
  }
}

REGISTER_DISPATCH(sin_stub, &sin_kernel_cuda);

} // namespace at::native
