#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <c10/util/BFloat16-math.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at::native {

void nextafter_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(kBFloat16, iter.common_dtype(), "nextafter_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return std::nextafter(a, b);
    });
  });
}

void heaviside_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBool, kBFloat16, iter.dtype(), "heaviside_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a == 0 ? b : static_cast<scalar_t>(a > 0);
    });
  });
}

REGISTER_DISPATCH(nextafter_stub, &nextafter_kernel_cuda);
REGISTER_DISPATCH(heaviside_stub, &heaviside_kernel_cuda);

} // namespace at::native
