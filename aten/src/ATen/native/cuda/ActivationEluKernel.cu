#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>

namespace at::native {
namespace {

void elu_kernel(
    TensorIteratorBase& iter,
    const Scalar& alpha,
    const Scalar& scale,
    const Scalar& input_scale) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "elu_cuda",
      [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        auto negcoef = alpha.to<opmath_t>() * scale.to<opmath_t>();
        auto poscoef = scale.to<opmath_t>();
        auto negiptcoef = input_scale.to<opmath_t>();
        gpu_kernel(
            iter,
            [negcoef, poscoef, negiptcoef] GPU_LAMBDA(scalar_t a) -> scalar_t {
              opmath_t aop = static_cast<opmath_t>(a);
              return aop > 0 ? aop * poscoef
                             : std::expm1(aop * negiptcoef) * negcoef;
            });
      });
}

void elu_backward_kernel(
    TensorIteratorBase& iter,
    const Scalar& alpha,
    const Scalar& scale,
    const Scalar& input_scale,
    bool is_result) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "elu_backward_cuda",
      [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        auto negcoef = alpha.to<opmath_t>() * scale.to<opmath_t>();
        auto poscoef = scale.to<opmath_t>();
        auto negiptcoef = input_scale.to<opmath_t>();
        gpu_kernel(
            iter,
            [negcoef, poscoef, negiptcoef, is_result] GPU_LAMBDA(
                scalar_t a, scalar_t b) -> scalar_t {
              opmath_t aop = static_cast<opmath_t>(a);
              opmath_t bop = static_cast<opmath_t>(b);

              if (is_result) {
                return bop <= 0 ? aop * negiptcoef * (bop + negcoef)
                                : aop * poscoef;
              } else {
                return bop <= 0
                    ? aop * negiptcoef * negcoef * std::exp(bop * negiptcoef)
                    : aop * poscoef;
              }
            });
      });
}
} // namespace

REGISTER_DISPATCH(elu_stub, &elu_kernel);
REGISTER_DISPATCH(elu_backward_stub, &elu_backward_kernel);

} // namespace at::native
