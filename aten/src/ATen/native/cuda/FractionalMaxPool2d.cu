#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/NumericUtils.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/native/FractionalMaxPooling.h>
#include <c10/macros/Macros.h>
#include <c10/util/Exception.h>
#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/fractional_max_pool2d_backward_native.h>
#include <ATen/ops/fractional_max_pool2d_native.h>
#endif

#include <algorithm>
#include <cfloat>
#include <cmath>

namespace at::native {

using namespace at::cuda::detail;

namespace {

template <typename scalar_t, typename accscalar_t>
__device__ inline int get_interval(accscalar_t sample,
  int index, int inputSize, int outputSize, int poolSize) {
  accscalar_t alpha = static_cast<accscalar_t>(inputSize - poolSize) /
    static_cast<accscalar_t>(outputSize - 1);
  if (index == outputSize - 1) {
    return inputSize - poolSize;
  } else {
    return static_cast<int>((index + sample) * alpha) -
      static_cast<int>(sample * alpha);
  }
}

template <typename scalar_t>
__global__ void fractional_max_pool2d_out_cuda_frame(
  PackedTensorAccessor<scalar_t, 4> output,
  PackedTensorAccessor<int64_t, 4> indices,
  PackedTensorAccessor<const scalar_t, 4> input,
  PackedTensorAccessor<const scalar_t, 3> samples,
  int poolSizeH, int poolSizeW) {

  using accscalar_t = at::acc_type<scalar_t, /*is_cuda=*/true>;

  int ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;

  // Each thread generates a specific output point
  if (ourOutputPoint < output.size(2) * output.size(3)) {
    int outputW = ourOutputPoint % output.size(3);
    int outputH = ourOutputPoint / output.size(3);

    int poolW = get_interval<scalar_t, accscalar_t>(
      static_cast<accscalar_t>(samples[batch][plane][0]),
        outputW, input.size(3), output.size(3), poolSizeW);
    int poolH = get_interval<scalar_t, accscalar_t>(
      static_cast<accscalar_t>(samples[batch][plane][1]),
        outputH, input.size(2), output.size(2), poolSizeH);

    scalar_t maxVal = at::numeric_limits<scalar_t>::lower_bound();
    int maxIndex = poolH * input.size(3) + poolW;

    for (int h = poolH; h < poolH + poolSizeH; ++h) {
      if (poolSizeW < 2 || poolSizeW > 7) {
        for (int w = poolW; w < poolW + poolSizeW; ++w) {
          scalar_t val = input[batch][plane][h][w];
          // for consistency with THNN, favor the first max
          if (val > maxVal || at::_isnan(val)) {
            maxIndex = h * input.size(3) + w;
            maxVal = val;
          }
        }
      } else {
        for (int i = 0; i < poolSizeW; ++i) {
          int w = i + poolW;
          scalar_t val = input[batch][plane][h][w];
          // for consistency with THNN, favor the first max
          if (val > maxVal || at::_isnan(val)) {
            maxIndex = h * input.size(3) + w;
            maxVal = val;
          }
        }
      }
    }

    indices[batch][plane][outputH][outputW] = maxIndex;
    output[batch][plane][outputH][outputW] = maxVal;
  }
}

template <typename scalar_t>
__global__ void fractional_max_pool2d_backward_out_cuda_frame(
  PackedTensorAccessor<scalar_t, 4> gradInput,
  PackedTensorAccessor<const scalar_t, 4> gradOutput,
  PackedTensorAccessor<const int64_t, 4> indices) {
  // Output (h, w) point that this thread is responsible for
  int ourOutputPoint = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;

  // Each thread generates a specific output point
  if (ourOutputPoint < gradOutput.size(2) *
    gradOutput.size(3)) {
    int outputW = ourOutputPoint % gradOutput.size(3);
    int outputH = ourOutputPoint / gradOutput.size(3);

    int index = indices[batch][plane][outputH][outputW];
    CUDA_KERNEL_ASSERT(index >= 0);
    int inputW = index % gradInput.size(3);
    int inputH = index / gradInput.size(3);
    CUDA_KERNEL_ASSERT(inputH < gradInput.size(2));

    gpuAtomicAddNoReturn(
      &gradInput[batch][plane][inputH][inputW],
      gradOutput[batch][plane][outputH][outputW]
    );
  }
}

} // anonymous namespace

TORCH_IMPL_FUNC(fractional_max_pool2d_out_cuda) (
  const Tensor& input,
  IntArrayRef pool_size,
  IntArrayRef output_size,
  const Tensor& randomSamples,
  const Tensor& output,
  const Tensor& indices
) {
  fractional_max_pool_check_shape</*ndim*/ 2>(input, randomSamples);

  int planeDim = 0;

  int ndims = input.ndimension();

  if (ndims == 4) {
    planeDim++;
  }

  /* sizes */
  int numPlanes = input.size(planeDim);

  int outputH = output_size[0];
  int outputW = output_size[1];
  int poolSizeH = pool_size[0];
  int poolSizeW = pool_size[1];

  auto output_ = output;
  auto input_ = input;
  auto indices_ = indices;

  if(ndims == 3) {
    output_ = output_.reshape({1, numPlanes, outputH, outputW});
    indices_ = indices_.reshape({1, numPlanes, outputH, outputW});
    input_ = input_.reshape({1, input.size(0), input.size(1), input.size(2)});
  }

  if (output_.numel() == 0) {
    return;
  }

  // block is limited to 4 warps
  // grid handles overflow per each plane
  int outputPlaneSize = output_.size(2) *
    output_.size(3);
  dim3 grid((outputPlaneSize + 127) / 128, // ceil(outputPlaneSize / 128)
            input_.size(1),
            input_.size(0));
  dim3 block(outputPlaneSize > 128 ? 128 : outputPlaneSize);

  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    input.scalar_type(),
    "fractional_max_pool2d_out_cuda_frame",
    [&] {
      auto devInput = input_.packed_accessor64<const scalar_t, 4>();
      auto devOutput = output_.packed_accessor64<scalar_t, 4>();
      auto devIndices = indices_.packed_accessor64<int64_t, 4>();
      auto devSamples = randomSamples.packed_accessor64<const scalar_t, 3>();
      fractional_max_pool2d_out_cuda_frame<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
          devOutput, devIndices, devInput, devSamples,
          poolSizeH, poolSizeW);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
     }
   );
}

TORCH_IMPL_FUNC(fractional_max_pool2d_backward_cuda)(
  const Tensor& gradOutput,
  const Tensor& input,
  IntArrayRef pool_size /* unused */,
  IntArrayRef output_size,
  const Tensor& indices,
  const Tensor& gradInput)
{

  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("fractional_max_pool2d_backward_cuda");

  int dimh = 1;
  int dimw = 2;

  int ndims = input.ndimension();
  if (ndims == 4) {
    dimh++;
    dimw++;
  }

  /* sizes */
  int inputH = input.size(dimh);
  int inputW = input.size(dimw);

  int outputH = output_size[0];
  int outputW = output_size[1];

  if (gradInput.numel() == 0) {
    return;
  }

  gradInput.zero_();

  auto gradInput_ = gradInput;
  auto gradOutput_ = gradOutput;
  auto indices_ = indices;

  if(ndims == 3) {
    gradInput_ = gradInput_.reshape({1, input.size(0), inputH, inputW});
    gradOutput_ = gradOutput_.reshape({1, gradOutput.size(0), outputH, outputW});
    indices_ = indices_.reshape({1, indices_.size(0), outputH, outputW});
  }

  /* backprop */
  // block is limited to 4 warps
  // grid handles overflow per each plane
  int outputPlaneSize = gradOutput_.size(2) *
    gradOutput_.size(3);
  dim3 grid((outputPlaneSize + 127) / 128, // ceil(outputPlaneSize / 128)
            gradInput_.size(1),
            gradInput_.size(0));
  dim3 block(outputPlaneSize > 128 ? 128 : outputPlaneSize);

  auto devIndices = indices_.packed_accessor64<const int64_t, 4>();
  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    gradOutput.scalar_type(),
    "fractional_max_pool2d_backward_out_cuda_frame",
    [&] {
      auto devGradInput = gradInput_.packed_accessor64<scalar_t, 4>();
      auto devGradOutput = gradOutput_.packed_accessor64<const scalar_t, 4>();
      fractional_max_pool2d_backward_out_cuda_frame<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
        devGradInput, devGradOutput, devIndices);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  );
}

}// at::native
