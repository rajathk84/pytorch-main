#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/UpSample.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/upsample_nearest1d_native.h>
#include <ATen/ops/upsample_nearest1d_backward_native.h>
#include <ATen/ops/_upsample_nearest_exact1d_native.h>
#include <ATen/ops/_upsample_nearest_exact1d_backward_native.h>
#endif

namespace at::native {
namespace {

#define MAX_THREADS 512

// Define a typedef to dispatch to nearest_neighbor_compute_source_index or
// nearest_neighbor_exact_compute_source_index
typedef int (*nn_compute_source_index_fn_t)(const float, int, int);

// Define a typedef to dispatch to nearest_neighbor_bw_compute_source_index or
// nearest_neighbor_exact_bw_compute_source_index
typedef int (*nn_bw_compute_source_index_fn_t)(const float, int, int);


// see NOTE [ Nearest neighbor upsampling kernel implementation ]
template <typename scalar_t, nn_compute_source_index_fn_t nn_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest1d_out_frame(
    const scalar_t* input,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_w,
    size_t dst_dim_w,
    scalar_t* output,
    float scale_factor) {
  int dst_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_w)
    return;

  int c = (dst_idx / dst_dim_w) % dim_c;

  int dst_x = dst_idx % dst_dim_w;
  int src_x = nn_compute_source_index_fn(scale_factor, dst_x, src_dim_w);

  int src_idx = c * src_dim_w + src_x;
  int src_stride = dim_c * src_dim_w;
  int dst_stride = dim_c * dst_dim_w;

  for (int b = 0; b < dim_b; b++) {
    output[dst_idx] = input[src_idx];
    src_idx += src_stride;
    dst_idx += dst_stride;
  }
}

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
// Backward operation
template <typename scalar_t, typename accscalar_t, nn_bw_compute_source_index_fn_t nn_bw_compute_source_index_fn>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest1d_backward_out_frame(
    const scalar_t* grad_o,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_w,
    size_t dst_dim_w,
    scalar_t* grad_i,
    float scale_factor) {

  int dst_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_w)
    return;

  int c = (dst_idx / (dst_dim_w)) % dim_c;

  int dst_x = dst_idx % dst_dim_w;
  // note that we do not want to clamp src_x to src_dim_w, since we might
  // intentionally want to skip in case of scale_factor < 1.0
  int src_x = nn_bw_compute_source_index_fn(scale_factor, dst_x, src_dim_w);
  int src_x_up = nn_bw_compute_source_index_fn(scale_factor, dst_x+1, src_dim_w);

  for (int b = 0; b < dim_b; b++) {
    accscalar_t grad = 0;
    int src_idx = b * dim_c * src_dim_w + c * src_dim_w + src_x;
    for (int x = src_x; x < src_x_up; x++) {
      grad += grad_o[src_idx++];
    }
    grad_i[dst_idx] = grad;
    dst_idx += dim_c * dst_dim_w;
  }
}

template<nn_compute_source_index_fn_t nn_compute_source_index_fn>
static void upsample_nearest1d_out_cuda_template(
    const Tensor& output,
    const Tensor& input_,
    IntArrayRef output_size,
    std::optional<double> scales) {
  TensorArg input_arg{input_, "input_", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_nearest1d_out_cuda", {input_arg, output_arg});

  int output_width = output_size[0];

  int nbatch = input_.size(0);
  int channels = input_.size(1);
  int input_width = input_.size(2);

  Tensor input = input_.contiguous();

  if (input.numel() == 0) {
    return;
  }

  // upsample_nearest1d meta call makes sure `nbatch != 0`
  unsigned int n = output.numel() / nbatch;
  dim3 bdim{std::min<unsigned int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
  dim3 gdim{ceil_div(n, bdim.x)};
  // safe check for int32 indexing; implicitly restrict launch config for kernel
  TORCH_CHECK(output.numel() <= std::numeric_limits<int32_t>::max());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte, input.scalar_type(), "upsample_nearest1d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.const_data_ptr<scalar_t>();
        auto odata = output.mutable_data_ptr<scalar_t>();

        const float scale_factor = compute_scales_value<float>(scales, input_width, output_width);

        upsample_nearest1d_out_frame<scalar_t, nn_compute_source_index_fn><<<gdim, bdim, 0, stream>>>(
            idata, nbatch, channels, input_width, output_width, odata, scale_factor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

template<nn_compute_source_index_fn_t nn_bw_compute_source_index_fn>
static void upsample_nearest1d_backward_out_cuda_template(
    const Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      "upsample_nearest1d_backward_out_cuda_template",
      {grad_output_arg, grad_input_arg});

  int output_width = output_size[0];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_width = input_size[2];

  Tensor grad_output = grad_output_.contiguous();

  if (grad_input.numel() == 0) {
    return;
  }

  // upsample_nearest1d meta call makes sure `nbatch != 0`
  unsigned int n = grad_input.numel() / nbatch;
  dim3 bdim{std::min<unsigned int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
  dim3 gdim{ceil_div(n, bdim.x)};
  // safe check for int32 indexing; implicitly restrict launch config for kernel
  TORCH_CHECK(grad_input.numel() <= std::numeric_limits<int32_t>::max(),
    "upsample_nearest1d_backward only supports input tensors with less than INT_MAX elements, but got ", grad_input.sizes());
  TORCH_CHECK(grad_output.numel() <= std::numeric_limits<int32_t>::max(),
        "upsample_nearest1d_backward only supports output tensors with less than INT_MAX elements, but got ", grad_output.sizes());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Byte, grad_output.scalar_type(), "upsample_nearest1d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.mutable_data_ptr<scalar_t>();
        auto odata = grad_output.const_data_ptr<scalar_t>();

        const float scale_factor = compute_scales_value_backwards<float>(scales, output_width, input_width);

        upsample_nearest1d_backward_out_frame<scalar_t, accscalar_t, nn_bw_compute_source_index_fn>
            <<<gdim, bdim, 0, stream>>>(
                odata, nbatch, channels, output_width, input_width, idata, scale_factor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

TORCH_IMPL_FUNC(upsample_nearest1d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    std::optional<double> scales,
    const Tensor& output
) {
  upsample_nearest1d_out_cuda_template<nearest_neighbor_compute_source_index>(
      output, input, output_size, scales);
}

TORCH_IMPL_FUNC(_upsample_nearest_exact1d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    std::optional<double> scales,
    const Tensor& output
) {
  upsample_nearest1d_out_cuda_template<nearest_neighbor_exact_compute_source_index>(output, input, output_size, scales);
}

TORCH_IMPL_FUNC(upsample_nearest1d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales,
    const Tensor& grad_input
) {
  upsample_nearest1d_backward_out_cuda_template<nearest_neighbor_bw_compute_source_index>(
      grad_input, grad_output, output_size, input_size, scales);
}

TORCH_IMPL_FUNC(_upsample_nearest_exact1d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    std::optional<double> scales,
    const Tensor& grad_input
) {
  upsample_nearest1d_backward_out_cuda_template<nearest_neighbor_exact_bw_compute_source_index>(
      grad_input, grad_output, output_size, input_size, scales);
}

} // namespace at::native
