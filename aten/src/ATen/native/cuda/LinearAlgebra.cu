#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/LinearAlgebra.h>
#include <ATen/native/BatchLinearAlgebra.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/native/ReduceOps.h>
#include <c10/core/Scalar.h>

#include <thrust/swap.h>

namespace at::native {

namespace {

void addr_kernel_cuda(TensorIterator &iter, const Scalar& beta, const Scalar& alpha) {
  if (iter.dtype() == ScalarType::Bool) {
    using scalar_t = bool;
    auto beta_val = beta.to<scalar_t>();
    auto alpha_val = alpha.to<scalar_t>();

    // when beta is false, values in self should be ignored,
    // nans and infs in self should not propagate.
    if (beta_val == false) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return alpha_val && vec1_val && vec2_val;
        }
      );
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return (beta_val && self_val) || (alpha_val && vec1_val && vec2_val);
        }
      );
    }
    return;
  }

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kBFloat16, kHalf,
                                         iter.dtype(), "addr_cuda", [&] {
    auto beta_val = beta.to<scalar_t>();
    auto alpha_val = alpha.to<scalar_t>();

    scalar_t zero_val(0);
    // when beta==0, values in self should be ignored,
    // nans and infs in self should not propagate.
    if (beta_val == zero_val) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return alpha_val * vec1_val * vec2_val;
        }
      );
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return beta_val * self_val + alpha_val * vec1_val * vec2_val;
        }
      );
    }
  });
}


template <int n_threads, int n_elems_per_thread, typename func_t>
C10_LAUNCH_BOUNDS_2(n_threads, n_elems_per_thread)
__global__ void _elementwise_kernel(int total_n_elems, func_t f) {
  constexpr int total_work_block = n_threads * n_elems_per_thread;
  int idx = total_work_block * blockIdx.x + threadIdx.x;

  #pragma unroll
  for (int i = 0; i < n_elems_per_thread; ++i) {
    if (idx < total_n_elems) {
      f(idx);
      idx += n_threads;
    }
  }
}

template <int n_threads, int n_elems_per_thread, typename func_t>
static void _launch_kernel(int total_n_elems, func_t f) {
  TORCH_INTERNAL_ASSERT(
    total_n_elems >= 0 && total_n_elems <= std::numeric_limits<int32_t>::max()
  );

  dim3 block(n_threads);
  constexpr int total_work_block = n_threads * n_elems_per_thread;
  dim3 grid((total_n_elems + total_work_block - 1) / total_work_block);

  auto stream = at::cuda::getCurrentCUDAStream();
  _elementwise_kernel<n_threads, n_elems_per_thread, func_t>
    <<<grid, block, 0, stream>>>(total_n_elems, f);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void unpack_pivots_cuda_kernel(TensorIterator& iter, const int64_t dim_size, const int64_t max_pivot) {
  if (iter.numel() == 0) {
    return;
  }

  if (!iter.can_use_32bit_indexing()) {
    for (auto& sub_iter : iter.with_32bit_indexing()) {
      unpack_pivots_cuda_kernel(sub_iter, dim_size, max_pivot);
    }
    return;
  }

  const auto offset_calculator = make_offset_calculator<2>(iter);

  const auto perm_ptr = reinterpret_cast<char*>(iter.data_ptr(0));
  const auto pivots_ptr = reinterpret_cast<const char*>(iter.data_ptr(1));

  auto loop = [=]C10_DEVICE(const int idx) {
    const auto offsets = offset_calculator.get(idx);

    int64_t* const __restrict__ perm_data = reinterpret_cast<int64_t*>(perm_ptr + offsets[0]);
    const int32_t* const __restrict__ pivots_data = reinterpret_cast<const int32_t*>(pivots_ptr + offsets[1]);

    // QUESTION: can we mix 64bit offsets with 32bit Iterator indexing?
    for (int64_t i = 0; i < dim_size; ++i) {
      thrust::swap(
        perm_data[i],
        perm_data[pivots_data[i] - 1]
      );
    }
  };

  _launch_kernel<num_threads(), thread_work_size()>(iter.numel(), loop);
}
} // anonymous namespace

REGISTER_DISPATCH(unpack_pivots_stub, &unpack_pivots_cuda_kernel);
REGISTER_DISPATCH(addr_stub, &addr_kernel_cuda);
} // namespace at::native
