#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if 0 && AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char acos_name[] = "acos_impl";
#endif
void acos_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
    // Disabled due to accuracy issues
#if 0 && AT_USE_JITERATOR()
    static const auto acos_string = jiterator_stringify(
        template <typename T> T acos_impl(T a) { return std::acos(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "acos_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/acos_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, acos_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "acos_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::acos(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "acos_cuda",
        [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::acos(a);
          });
        });
  }
}

REGISTER_DISPATCH(acos_stub, &acos_kernel_cuda);

} // namespace at::native
