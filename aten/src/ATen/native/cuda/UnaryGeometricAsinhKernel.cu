#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if 0 && AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char asinh_name[] = "asinh_impl";
#endif

void asinh_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
    // Disabled due to accuracy issues
#if 0 && AT_USE_JITERATOR()
    static const auto asinh_string = jiterator_stringify(
        template <typename T> T asinh_impl(T a) { return std::asinh(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "asinh_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/asinh_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, asinh_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "asinh_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::asinh(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "asinh_cuda",
        [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::asinh(a);
          });
        });
  }
}

REGISTER_DISPATCH(asinh_stub, &asinh_kernel_cuda);

} // namespace at::native
