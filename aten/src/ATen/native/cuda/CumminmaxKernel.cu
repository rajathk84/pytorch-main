#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/core/TensorBase.h>
#include <ATen/Dispatch.h>

#include <ATen/native/cuda/ScanKernels.h>
#include <ATen/native/cuda/ScanUtils.cuh>

#include <limits>
#include <functional>

namespace at::native {

void launch_cummax_cuda_kernel(const TensorBase& self, const TensorBase& values, const TensorBase& indices, int64_t dim) {
  AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Bool, at::ScalarType::Half, at::ScalarType::BFloat16,
    self.scalar_type(), "cummax_cuda", [&]() {
    scalar_t init = self.is_floating_point() ? (-1*std::numeric_limits<scalar_t>::infinity()) : std::numeric_limits<scalar_t>::lowest();
    scan_dim_with_indices<scalar_t>(self, values, indices, dim, init, std::greater_equal<scalar_t>());
  });
}

void launch_cummin_cuda_kernel(const TensorBase& self, const TensorBase& values, const TensorBase& indices, int64_t dim) {
  AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Bool, at::ScalarType::Half, at::ScalarType::BFloat16,
    self.scalar_type(), "cummin_cuda", [&]() {
    scalar_t init = self.is_floating_point() ? std::numeric_limits<scalar_t>::infinity() : std::numeric_limits<scalar_t>::max();
    scan_dim_with_indices<scalar_t>(self, values, indices, dim, init, std::less_equal<scalar_t>());
  });
}

} // namespace at::native
