#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if 0 && AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char acosh_name[] = "acosh_impl";
#endif

void acosh_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if(at::isComplexType(common_dtype)) {
    // Disabled due to accuracy issues
#if 0 && AT_USE_JITERATOR()
  static const auto acosh_string = jiterator_stringify(
    template <typename T>
    T acosh_impl(T a) {
        return std::acosh(a);
    }
  );
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "acosh_name", [&]() {
    jitted_gpu_kernel<
        /*name=*/ acosh_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 1>(iter, acosh_string);
  });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "acosh_name", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        using opmath_t = at::opmath_type<scalar_t>;
        return ::acosh(static_cast<opmath_t>(a));
    });
  });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      common_dtype, "acosh_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::acosh(a);
        });
      });
  }
}

REGISTER_DISPATCH(acosh_stub, &acosh_kernel_cuda);

} // namespace at::native
