#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if 0 && AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char asin_name[] = "asin_impl";
#endif

void asin_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
    // Disabled due to accuracy issues
#if 0 && AT_USE_JITERATOR()
    static const auto asin_string = jiterator_stringify(
        template <typename T> T asin_impl(T a) { return std::asin(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "asin_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/asin_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, asin_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "asin_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::asin(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        kHalf, kBFloat16, common_dtype, "asin_cuda", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::asin(a);
          });
        });
  }
}

REGISTER_DISPATCH(asin_stub, &asin_kernel_cuda);

} // namespace at::native
