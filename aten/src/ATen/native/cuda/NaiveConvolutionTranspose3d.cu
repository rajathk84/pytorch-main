#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>

#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/HIPContext.h>

#include <ATen/native/ConvUtils.h>
#include <ATen/native/cuda/vol2col.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/sum.h>
#include <ATen/ops/ones.h>
#include <ATen/ops/slow_conv_transpose3d_native.h>
#endif

namespace at::native {
namespace {

static inline void slow_conv_transpose3d_shape_check(
    const Tensor& input,
    const Tensor& grad_output,
    const Tensor& weight,
    const Tensor& bias,
    int kernel_depth,
    int kernel_width,
    int kernel_height,
    int stride_depth,
    int stride_width,
    int stride_height,
    int padding_depth,
    int padding_width,
    int padding_height,
    int dilation_depth,
    int dilation_width,
    int dilation_height,
    int output_padding_depth,
    int output_padding_width,
    int output_padding_height,
    int weight_nullable) {
  TORCH_CHECK(
      input.numel() != 0 && (input.dim() == 4 || input.dim() == 5),
      "non-empty 4D or 5D (batch mode) tensor expected for input, but got: ",
      input.sizes());
  TORCH_CHECK(
      stride_depth > 0 && stride_width > 0 && stride_height > 0,
      "stride should be greater than zero, but got stride_depth: ",
      stride_depth,
      " stride_height: ",
      stride_height,
      " stride_width: ",
      stride_width);
  TORCH_CHECK(
      dilation_depth > 0 && dilation_width > 0 && dilation_height > 0,
      "dilation should be greater than zero, but got dilation_depth: ",
      dilation_depth,
      ", dilation_height: ",
      dilation_height,
      ", dilation_width: ",
      dilation_width);
  TORCH_CHECK(
      (output_padding_depth < stride_depth ||
       output_padding_depth < dilation_depth) &&
          (output_padding_width < stride_width ||
           output_padding_width < dilation_width) &&
          (output_padding_height < stride_height ||
           output_padding_height < dilation_height),
      "output padding must be smaller than either stride or dilation,",
      " but got output_padding_depth: ",
      output_padding_depth,
      " output_padding_height: ",
      output_padding_height,
      " output_padding_width: ",
      output_padding_width,
      " stride_depth: ",
      stride_depth,
      " stride_height: ",
      stride_height,
      " stride_width: ",
      stride_width,
      " dilation_depth: ",
      dilation_depth,
      " dilation_height: ",
      dilation_height,
      " dilation_width: ",
      dilation_width);

  // number of input & output planes and kernel size is indirectly defined by
  // the weight tensor
  if (weight.defined()) {
    TORCH_CHECK(
        weight.numel() != 0 && weight.dim() == 5,
        "non-empty 5D (n_output_plane x n_input_plane ",
        "x kernel_depth x kernel_height x kernel_width) tensor ",
        "expected for weight, but got: ",
        weight.sizes());
    if (bias.defined()) {
      check_dim_size(bias, 1, 0, weight.size(1));
    }
  } else if (!weight_nullable) {
    AT_ERROR("weight tensor is expected to be non-nullable");
  }

  int ndim = input.dim();
  int dimf = 0;
  int dimd = 1;
  int dimh = 2;
  int dimw = 3;

  if (ndim == 5) {
    dimf++;
    dimd++;
    dimh++;
    dimw++;
  }

  if (weight.defined()) {
    const int64_t n_input_plane = weight.size(0);
    check_dim_size(input, ndim, dimf, n_input_plane);
  }

  int64_t input_width = input.size(dimw);
  int64_t input_height = input.size(dimh);
  int64_t input_depth = input.size(dimd);

  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  if (output_depth < 1 || output_width < 1 || output_height < 1) {
    AT_ERROR(
        "Given input size per channel: (",
        input_depth,
        " x ",
        input_height,
        " x ",
        input_width,
        "). Calculated output size per channel: (",
        output_depth,
        " x ",
        output_height,
        " x ",
        output_width,
        "). Output size is too small");
  }

  if (grad_output.defined()) {
    if (weight.defined()) {
      const int64_t n_output_plane = weight.size(1);
      check_dim_size(grad_output, ndim, dimf, n_output_plane);
    } else if (bias.defined()) {
      const int64_t n_output_plane = bias.size(0);
      check_dim_size(grad_output, ndim, dimf, n_output_plane);
    }
    check_dim_size(grad_output, ndim, dimd, output_depth);
    check_dim_size(grad_output, ndim, dimh, output_height);
    check_dim_size(grad_output, ndim, dimw, output_width);
  }
}

void slow_conv_transpose3d_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    const Tensor& weight_,
    IntArrayRef kernel_size,
    const Tensor& bias_,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  TORCH_CHECK(
      kernel_size.size() == 3,
      "It is expected kernel_size equals to 3, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 3,
      "It is expected dilation equals to 3, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 3,
      "It is expected padding equals to 3, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 3,
      "It is expected stride equals to 3, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 3,
      "It is expected stride equals to 3, but got size ",
      output_padding.size());

  int64_t kernel_depth = kernel_size[0];
  int64_t kernel_height = kernel_size[1];
  int64_t kernel_width = kernel_size[2];
  int64_t dilation_depth = dilation[0];
  int64_t dilation_height = dilation[1];
  int64_t dilation_width = dilation[2];
  int64_t padding_depth = padding[0];
  int64_t padding_height = padding[1];
  int64_t padding_width = padding[2];
  int64_t stride_depth = stride[0];
  int64_t stride_height = stride[1];
  int64_t stride_width = stride[2];
  int64_t output_padding_depth = output_padding[0];
  int64_t output_padding_height = output_padding[1];
  int64_t output_padding_width = output_padding[2];

  int n_input_plane = weight_.size(0);
  int n_output_plane = weight_.size(1);

  TensorArg input_arg{input_, "input", 1}, output_arg{output, "output", 2},
      weight_arg{weight_, "weight", 3}, bias_arg{bias_, "bias", 4};

  checkAllSameGPU(
      "slow_conv_transpose3d_out_cuda",
      {input_arg, output_arg, weight_arg, bias_arg});

  slow_conv_transpose3d_shape_check(
      input_,
      Tensor(),
      weight_,
      bias_,
      kernel_depth,
      kernel_width,
      kernel_height,
      stride_depth,
      stride_width,
      stride_height,
      padding_depth,
      padding_width,
      padding_height,
      dilation_depth,
      dilation_width,
      dilation_height,
      output_padding_depth,
      output_padding_width,
      output_padding_height,
      0);

  Tensor input = input_.contiguous();
  Tensor weight = weight_.contiguous();
  Tensor bias = bias_.defined() ? bias_.contiguous() : bias_;

  int is_batch = false;
  if (input.dim() == 4) {
    // Force batch
    is_batch = true;
    input.resize_(
        {1, input.size(0), input.size(1), input.size(2), input.size(3)});
  }

  int64_t input_width = input.size(4);
  int64_t input_height = input.size(3);
  int64_t input_depth = input.size(2);

  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Resize output
  output.resize_(
      {batch_size, n_output_plane, output_depth, output_height, output_width});

  // Create temporary columns
  Tensor columns = at::empty({n_output_plane * kernel_width * kernel_height * kernel_depth,
      input_depth * input_height * input_width}, input.options());

  // Define a buffer of ones, for bias accumulation
  Tensor ones = bias.defined() ? at::ones({output_depth, output_height, output_width}, input_.options()) : Tensor();

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "slow_conv_transpose3d_out_cuda", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        // Helpers
        Tensor input_n;
        Tensor output_n;

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix multiply per output:
          input_n = input.select(0, elt);
          output_n = output.select(0, elt);

          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m =
              weight.size(1) * weight.size(2) * weight.size(3) * weight.size(4);
          int64_t n = columns.size(1);
          int64_t k = weight.size(0);

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          at::cuda::blas::gemm<scalar_t>(
              'n',
              't',
              n,
              m,
              k,
              static_cast<scalar_t>(1),
              input_n.const_data_ptr<scalar_t>(),
              n,
              weight.const_data_ptr<scalar_t>(),
              m,
              static_cast<scalar_t>(0),
              columns.mutable_data_ptr<scalar_t>(),
              n);

          // Unpack columns back into input:
          at::native::col2vol<scalar_t, accscalar_t>(
              at::cuda::getCurrentCUDAStream(),
              columns.const_data_ptr<scalar_t>(),
              n_output_plane,
              output_depth,
              output_height,
              output_width,
              input_depth,
              input_height,
              input_width,
              kernel_depth,
              kernel_height,
              kernel_width,
              padding_depth,
              padding_height,
              padding_width,
              stride_depth,
              stride_height,
              stride_width,
              dilation_depth,
              dilation_height,
              dilation_width,
              output_n.mutable_data_ptr<scalar_t>());

          // Do Bias after:
          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m_ = n_output_plane;
          int64_t n_ = output_depth * output_height * output_width;
          int64_t k_ = 1;

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          if (bias.defined()) {
            at::cuda::blas::gemm<scalar_t>(
                't',
                'n',
                n_,
                m_,
                k_,
                static_cast<scalar_t>(1),
                ones.const_data_ptr<scalar_t>(),
                k_,
                bias.const_data_ptr<scalar_t>(),
                k_,
                static_cast<scalar_t>(1),
                output_n.mutable_data_ptr<scalar_t>(),
                n_);
          }
        }

        // Resize output
        if (is_batch) {
          output.resize_(
              {n_output_plane, output_depth, output_height, output_width});
          input.resize_(
              {n_input_plane, input_depth, input_height, input_width});
        }
      });
}

void slow_conv_transpose3d_backward_out_cuda_template(
    const Tensor& input_,
    const Tensor& grad_output_,
    Tensor& grad_input,
    const Tensor& weight_,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  TORCH_CHECK(
      kernel_size.size() == 3,
      "It is expected kernel_size equals to 3, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 3,
      "It is expected dilation equals to 3, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 3,
      "It is expected padding equals to 3, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 3,
      "It is expected stride equals to 3, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 3,
      "It is expected stride equals to 3, but got size ",
      output_padding.size());

  int n_input_plane = weight_.size(0);
  int n_output_plane = weight_.size(1);

  int64_t kernel_depth = kernel_size[0];
  int64_t kernel_height = kernel_size[1];
  int64_t kernel_width = kernel_size[2];
  int64_t dilation_depth = dilation[0];
  int64_t dilation_height = dilation[1];
  int64_t dilation_width = dilation[2];
  int64_t padding_depth = padding[0];
  int64_t padding_height = padding[1];
  int64_t padding_width = padding[2];
  int64_t stride_depth = stride[0];
  int64_t stride_height = stride[1];
  int64_t stride_width = stride[2];
  int64_t output_padding_depth = output_padding[0];
  int64_t output_padding_height = output_padding[1];
  int64_t output_padding_width = output_padding[2];

  TensorArg input_arg{input_, "input", 1},
      grad_output_arg{grad_output_, "grad_output", 2},
      weight_arg{weight_, "weight", 3},
      grad_input_arg{grad_input, "grad_input", 4};

  checkAllSameGPU(
      "slow_conv_transpose3d_backward_out_cuda",
      {input_arg,
       grad_output_arg,
       weight_arg,
       grad_input_arg});

  slow_conv_transpose3d_shape_check(
      input_,
      grad_output_,
      weight_,
      Tensor(),
      kernel_depth,
      kernel_width,
      kernel_height,
      stride_depth,
      stride_width,
      stride_height,
      padding_depth,
      padding_width,
      padding_height,
      dilation_depth,
      dilation_width,
      dilation_height,
      output_padding_depth,
      output_padding_width,
      output_padding_height,
      0);

  Tensor input = input_.contiguous();
  Tensor grad_output = grad_output_.contiguous();
  Tensor weight = weight_.contiguous();

  bool is_batch = false;
  if (input.dim() == 4) {
    // Force batch
    is_batch = true;
    input.resize_(
        {1, input.size(0), input.size(1), input.size(2), input.size(3)});
    grad_output.resize_({1,
                         grad_output.size(0),
                         grad_output.size(1),
                         grad_output.size(2),
                         grad_output.size(3)});
  }

  int64_t input_width = input.size(4);
  int64_t input_height = input.size(3);
  int64_t input_depth = input.size(2);
  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Resize output
  grad_input.resize_(
      {batch_size, n_input_plane, input_depth, input_height, input_width});

  // Create temporary columns
  bool need_columns = (kernel_depth != 1 || kernel_height != 1 || kernel_width != 1 ||
      stride_depth != 1 || stride_height != 1 || stride_width != 1 ||
      dilation_depth != 1 || dilation_height != 1 ||
      dilation_width != 1 || padding_depth != 0 ||
      padding_height != 0 || padding_width != 0);
  Tensor grad_columns = need_columns ? at::empty({n_output_plane * kernel_width * kernel_height * kernel_depth,
      input_depth * input_height * input_width}, input.options()) : Tensor();

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "slow_conv_transpose3d_backward_out_cuda", [&] {
        // Helpers
        Tensor grad_input_n;
        Tensor grad_output_n;

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix multiply per sample:
          grad_input_n = grad_input.select(0, elt);
          grad_output_n = grad_output.select(0, elt);

          if (need_columns) {
            // Extract columns:
            at::native::vol2col<scalar_t>(
                at::cuda::getCurrentCUDAStream(),
                grad_output_n.const_data_ptr<scalar_t>(),
                n_output_plane,
                output_depth,
                output_height,
                output_width,
                input_depth,
                input_height,
                input_width,
                kernel_depth,
                kernel_height,
                kernel_width,
                padding_depth,
                padding_height,
                padding_width,
                stride_depth,
                stride_height,
                stride_width,
                dilation_depth,
                dilation_height,
                dilation_width,
                grad_columns.mutable_data_ptr<scalar_t>());
          }

          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m = weight.size(0);
          int64_t n = input_depth * input_height * input_width;
          int64_t k =
              weight.size(1) * weight.size(2) * weight.size(3) * weight.size(4);

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          auto gemm_in_ptr = need_columns ? grad_columns.const_data_ptr<scalar_t>()
              : grad_output_n.const_data_ptr<scalar_t>();
          at::cuda::blas::gemm<scalar_t>(
              'n',
              'n',
              n,
              m,
              k,
              static_cast<scalar_t>(1),
              gemm_in_ptr,
              n,
              weight.const_data_ptr<scalar_t>(),
              k,
              static_cast<scalar_t>(0),
              grad_input_n.mutable_data_ptr<scalar_t>(),
              n);
        }

        // Resize output
        if (is_batch) {
          grad_output.resize_(
              {n_output_plane, output_depth, output_height, output_width});
          input.resize_(
              {n_input_plane, input_depth, input_height, input_width});
          grad_input.resize_(
              {n_input_plane, input_depth, input_height, input_width});
        }
      });
}

void slow_conv_transpose3d_acc_grad_parameters_cuda(
    const Tensor& input_,
    const Tensor& grad_output_,
    Tensor& grad_weight,
    Tensor& grad_bias,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    int scale_) {
  TORCH_CHECK(
      kernel_size.size() == 3,
      "It is expected kernel_size equals to 3, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 3,
      "It is expected dilation equals to 3, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 3,
      "It is expected padding equals to 3, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 3,
      "It is expected stride equals to 3, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 3,
      "It is expected stride equals to 3, but got size ",
      output_padding.size());

  int64_t kernel_depth = kernel_size[0];
  int64_t kernel_height = kernel_size[1];
  int64_t kernel_width = kernel_size[2];
  int64_t dilation_depth = dilation[0];
  int64_t dilation_height = dilation[1];
  int64_t dilation_width = dilation[2];
  int64_t padding_depth = padding[0];
  int64_t padding_height = padding[1];
  int64_t padding_width = padding[2];
  int64_t stride_depth = stride[0];
  int64_t stride_height = stride[1];
  int64_t stride_width = stride[2];
  int64_t output_padding_depth = output_padding[0];
  int64_t output_padding_height = output_padding[1];
  int64_t output_padding_width = output_padding[2];

  TensorArg input_arg{input_, "input", 1},
      grad_output_arg{grad_output_, "grad_output", 2},
      grad_weight_arg{grad_weight, "grad_weight", 3},
      grad_bias_arg{grad_bias, "grad_bias", 4};

  checkAllSameGPU(
      "slow_conv_transpose3d_acc_grad_parameters_cuda",
      {input_arg,
       grad_output_arg,
       grad_weight_arg,
       grad_bias_arg});

  slow_conv_transpose3d_shape_check(
      input_,
      grad_output_,
      grad_weight,
      grad_bias,
      kernel_depth,
      kernel_width,
      kernel_height,
      stride_depth,
      stride_width,
      stride_height,
      padding_depth,
      padding_width,
      padding_height,
      dilation_depth,
      dilation_width,
      dilation_height,
      output_padding_depth,
      output_padding_width,
      output_padding_height,
      1);

  int n_output_plane;
  if (grad_weight.defined()) {
    n_output_plane = grad_weight.size(1);
  } else if (grad_bias.defined()) {
    n_output_plane = grad_bias.size(0);
  } else {
    return;
  }

  if (grad_weight.defined()) {
    TORCH_CHECK(
        grad_weight.is_contiguous(), "grad_weight needs to be contiguous");
  }
  if (grad_bias.defined()) {
    TORCH_CHECK(grad_bias.is_contiguous(), "grad_bias needs to be contiguous");
  }

  Tensor input = input_.contiguous();
  Tensor grad_output = grad_output_.contiguous();

  bool is_batch = false;
  if (input.dim() == 4) {
    // Force batch
    is_batch = true;
    input.resize_(
        {1, input.size(0), input.size(1), input.size(2), input.size(3)});
    grad_output.resize_({1,
                         grad_output.size(0),
                         grad_output.size(1),
                         grad_output.size(2),
                         grad_output.size(3)});
  }

  int64_t input_width = input.size(4);
  int64_t input_height = input.size(3);
  int64_t input_depth = input.size(2);

  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Create temporary columns
  bool need_columns = (kernel_depth != 1 || kernel_height != 1 || kernel_width != 1 ||
      stride_depth != 1 || stride_height != 1 || stride_width != 1 ||
      dilation_depth != 1 || dilation_height != 1 ||
      dilation_width != 1 || padding_depth != 0 ||
      padding_height != 0 || padding_width != 0);
  Tensor columns = need_columns ? at::empty({n_output_plane * kernel_width * kernel_height * kernel_depth,
      input_depth * input_height * input_width}, input.options()) : Tensor();

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(),
      "slow_conv_transpose3d_acc_grad_parameters_cuda",
      [&] {
        // Helpers
        Tensor input_n;
        Tensor grad_output_n;

        scalar_t scale = static_cast<scalar_t>(scale_);

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix multiply per output:
          grad_output_n = grad_output.select(0, elt);

          // Do Weight:
          if (grad_weight.defined()) {
            // Matrix multiply per output:
            input_n = input.select(0, elt);

            if (need_columns) {
              // Extract columns:
              at::native::vol2col<scalar_t>(
                  at::cuda::getCurrentCUDAStream(),
                  grad_output_n.const_data_ptr<scalar_t>(),
                  n_output_plane,
                  output_depth,
                  output_height,
                  output_width,
                  input_depth,
                  input_height,
                  input_width,
                  kernel_depth,
                  kernel_height,
                  kernel_width,
                  padding_depth,
                  padding_height,
                  padding_width,
                  stride_depth,
                  stride_height,
                  stride_width,
                  dilation_depth,
                  dilation_height,
                  dilation_width,
                  columns.mutable_data_ptr<scalar_t>());
            }

            // M,N,K are dims of matrix A and B
            // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
            int64_t n = n_output_plane * kernel_width * kernel_height * kernel_depth;
            int64_t m = input_n.size(0); // n_input_plane
            int64_t k = input_depth * input_height * input_width;

            // Do GEMM (note: this is a bit confusing because gemm assumes
            // column-major matrices)
            auto gemm_in_ptr = need_columns ? columns.const_data_ptr<scalar_t>() : grad_output_n.const_data_ptr<scalar_t>();
            at::cuda::blas::gemm<scalar_t>(
                't',
                'n',
                n,
                m,
                k,
                scale,
                gemm_in_ptr,
                k,
                input_n.const_data_ptr<scalar_t>(),
                k,
                static_cast<scalar_t>(1),
                grad_weight.mutable_data_ptr<scalar_t>(),
                n);
          }
        }

        if (grad_bias.defined()) {
          at::sum_out(grad_bias, grad_output, IntArrayRef{0, 2, 3, 4});
        }

        // Resize
        if (is_batch) {
          grad_output.resize_(
              {n_output_plane, output_depth, output_height, output_width});
          input.resize_(
              {input.size(1), input_depth, input_height, input_width});
        }
      });
}

} // namespace

Tensor& slow_conv_transpose3d_out_cuda(const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size, const std::optional<Tensor>& bias_opt,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    Tensor& output) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> bias_maybe_owned = at::borrow_from_optional_tensor(bias_opt);
  const Tensor& bias = *bias_maybe_owned;

  slow_conv_transpose3d_out_cuda_template(
      output,
      input,
      weight,
      kernel_size,
      bias,
      stride,
      padding,
      output_padding,
      dilation);

  return output;
}

Tensor slow_conv_transpose3d_cuda(
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size, const std::optional<Tensor>& bias_opt,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> bias_maybe_owned = at::borrow_from_optional_tensor(bias_opt);
  const Tensor& bias = *bias_maybe_owned;

  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  slow_conv_transpose3d_out_cuda_template(
      output,
      input,
      weight,
      kernel_size,
      bias,
      stride,
      padding,
      output_padding,
      dilation);

  return output;
}

std::tuple<Tensor&, Tensor&, Tensor&> slow_conv_transpose3d_backward_out_cuda(const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    Tensor& grad_input,
    Tensor& grad_weight,
    Tensor& grad_bias) {
  if (grad_input.defined()) {
    slow_conv_transpose3d_backward_out_cuda_template(
        input,
        grad_output,
        grad_input,
        weight,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation);
  }

  if (grad_weight.defined()) {
    grad_weight.resize_(weight.sizes());
    grad_weight.zero_();
  }

  if (grad_bias.defined()) {
    grad_bias.resize_({weight.size(1)});
    grad_bias.zero_();
  }

  if (grad_weight.defined() || grad_bias.defined()) {
    slow_conv_transpose3d_acc_grad_parameters_cuda(
        input,
        grad_output,
        grad_weight,
        grad_bias,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation,
        1);
  }

  return std::tuple<Tensor&, Tensor&, Tensor&>(
      grad_input, grad_weight, grad_bias);
}

std::tuple<Tensor, Tensor, Tensor> slow_conv_transpose3d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    std::array<bool, 3> output_mask) {
  Tensor grad_input;
  Tensor grad_weight;
  Tensor grad_bias;

  if (output_mask[0]) {
    grad_input = at::empty({0}, grad_output.options());
  } else {
    grad_input = Tensor();
  }

  if (output_mask[1]) {
    grad_weight = at::empty({0}, grad_output.options());
  } else {
    grad_weight = Tensor();
  }

  if (output_mask[2]) {
    grad_bias = at::empty({0}, grad_output.options());
  } else {
    grad_bias = Tensor();
  }

  if (grad_input.defined()) {
    slow_conv_transpose3d_backward_out_cuda_template(
        input,
        grad_output,
        grad_input,
        weight,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation);
  }

  if (grad_weight.defined()) {
    grad_weight.resize_(weight.sizes());
    grad_weight.zero_();
  }

  if (grad_bias.defined()) {
    grad_bias.resize_({weight.size(1)});
    grad_bias.zero_();
  }

  if (grad_weight.defined() || grad_bias.defined()) {
    slow_conv_transpose3d_acc_grad_parameters_cuda(
        input,
        grad_output,
        grad_weight,
        grad_bias,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation,
        1);
  }

  return std::tuple<Tensor, Tensor, Tensor>(grad_input, grad_weight, grad_bias);
}

REGISTER_CUDA_DISPATCH(slow_conv_transpose3d_backward_stub, &slow_conv_transpose3d_backward_cuda);

} // namespace at::native
