#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

namespace at::native {

void cauchy_kernel(TensorIteratorBase& iter, double median, double sigma, std::optional<Generator> gen) {
  auto generator = get_generator_or_default<CUDAGeneratorImpl>(gen, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::cauchy_kernel(iter, median, sigma, generator);
}

REGISTER_DISPATCH(cauchy_stub, &cauchy_kernel);

} // namespace at::native
