#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <limits>

namespace at::native {

#if AT_USE_JITERATOR()
CONSTEXPR_EXCEPT_WIN_CUDA char atanh_name[] = "atanh_impl";
#endif

void atanh_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR()
    static const auto atanh_string = jiterator_stringify(
        template <typename T> T atanh_impl(T a) { return std::atanh(a); });
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "atanh_name", [&]() {
          jitted_gpu_kernel<
              /*name=*/atanh_name,
              /*return_dtype=*/scalar_t,
              /*common_dtype=*/scalar_t,
              /*arity=*/1>(iter, atanh_string);
        });
#else
    AT_DISPATCH_COMPLEX_TYPES_AND(
        kComplexHalf, common_dtype, "atanh_name", [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return ::atanh(static_cast<opmath_t>(a));
          });
        });
#endif
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        ScalarType::Half,
        ScalarType::BFloat16,
        common_dtype,
        "atanh_cuda",
        [&]() {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::atanh(a);
          });
        });
  }
}

REGISTER_DISPATCH(atanh_stub, &atanh_kernel_cuda);

} // namespace at::native
