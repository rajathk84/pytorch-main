#include <gtest/gtest.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <optional>

#include <assert.h>

using namespace at;

// optional in cuda files
TEST(OptionalTest, OptionalTestCUDA) {
  if (!at::cuda::is_available()) return;
  std::optional<int64_t> trivially_destructible;
  std::optional<std::vector<int64_t>> non_trivially_destructible;
  ASSERT_FALSE(trivially_destructible.has_value());
  ASSERT_FALSE(non_trivially_destructible.has_value());

  trivially_destructible = {5};
  non_trivially_destructible = std::vector<int64_t>{5, 10};
  ASSERT_TRUE(trivially_destructible.has_value());
  ASSERT_TRUE(non_trivially_destructible.has_value());
}
