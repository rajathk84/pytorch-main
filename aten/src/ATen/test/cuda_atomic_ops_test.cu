#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/test/util/Macros.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>

#include <cmath>

constexpr int blocksize = 256;
constexpr int factor = 4;
constexpr int arraysize = blocksize / factor;

template <typename T>
__global__ void addition_test_kernel(T * a, T * sum) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = (tid) % arraysize;

  gpuAtomicAdd(&sum[idx], a[idx]);
}

template <typename T>
__global__ void mul_test_kernel(T * a, T * sum) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = (tid) % arraysize;

  gpuAtomicMul(&sum[idx], a[idx]);
}

template <typename T>
__global__ void max_test_kernel(T * a, T * max) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int a_idx = (tid) % (arraysize * factor);
  int idx = a_idx / factor;

  gpuAtomicMax(&max[idx], a[a_idx]);
}

template <typename T>
__global__ void min_test_kernel(T * a, T * min) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int a_idx = (tid) % (arraysize * factor);
  int idx = a_idx / factor;

  gpuAtomicMin(&min[idx], a[a_idx]);
}

template <typename T>
void test_atomic_add() {
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  T *ad, *sumd;

  std::vector<T> a(arraysize);
  std::vector<T> sum(arraysize);
  std::vector<T> answer(arraysize);

  for (int i = 0; i < arraysize; ++i) {
    a[i] = 1;
    sum[i] = 0;
    answer[i] = factor;
  }

  hipMalloc((void**)&ad, arraysize * sizeof(T));
  hipMalloc((void**)&sumd, arraysize * sizeof(T));

  hipMemcpy(ad, a.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(sumd, sum.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);

  addition_test_kernel<<<dimGrid, dimBlock>>>(ad, sumd);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipMemcpy(sum.data(), sumd, arraysize * sizeof(T), hipMemcpyDeviceToHost);

  for (int i = 0; i < arraysize; ++i) {
    ASSERT_EQ(sum[i], answer[i]) << typeid(T).name();
  }

  hipFree(ad);
  hipFree(sumd);
}

template <typename T>
void test_atomic_mul() {
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  T *ad, *sumd;

  std::vector<T> a(arraysize);
  std::vector<T> sum(arraysize);
  std::vector<T> answer(arraysize);

  for (int i = 0; i < arraysize; ++i) {
    a[i] = 2;
    sum[i] = 2;
    answer[i] = pow(sum[i], static_cast<T>(factor + 1));
  }

  hipMalloc((void**)&ad, arraysize * sizeof(T));
  hipMalloc((void**)&sumd, arraysize * sizeof(T));

  hipMemcpy(ad, a.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(sumd, sum.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);

  mul_test_kernel<<<dimGrid, dimBlock>>>(ad, sumd);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipMemcpy(sum.data(), sumd, arraysize * sizeof(T), hipMemcpyDeviceToHost);

  for (int i = 0; i < arraysize; ++i) {
    ASSERT_EQ(sum[i], answer[i]) << typeid(T).name();
  }

  hipFree(ad);
  hipFree(sumd);
}

template <typename T>
void test_atomic_max() {
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  T *ad, *sumd;

  std::vector<T> a(arraysize * factor);
  std::vector<T> sum(arraysize);
  std::vector<T> answer(arraysize);

  int j;
  for (int i = 0; i < arraysize * factor; ++i) {
    a[i] = i;
    if (i % factor == 0) {
      j = i / factor;
      sum[j] = std::numeric_limits<T>::lowest();
      answer[j] = (j + 1) * factor - 1;
    }
  }

  hipMalloc((void**)&ad, arraysize * factor * sizeof(T));
  hipMalloc((void**)&sumd, arraysize * sizeof(T));

  hipMemcpy(ad, a.data(), arraysize * factor * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(sumd, sum.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);

  max_test_kernel<<<dimGrid, dimBlock>>>(ad, sumd);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipMemcpy(sum.data(), sumd, arraysize * sizeof(T), hipMemcpyDeviceToHost);

  for (int i = 0; i < arraysize; ++i) {
    ASSERT_EQ(sum[i], answer[i]) << typeid(T).name();
  }

  hipFree(ad);
  hipFree(sumd);
}

template <typename T>
void test_atomic_min() {
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  T *ad, *sumd;

  std::vector<T> a(arraysize * factor);
  std::vector<T> sum(arraysize);
  std::vector<T> answer(arraysize);

  int j;
  for (int i = 0; i < arraysize * factor; ++i) {
    a[i] = i;
    if (i % factor == 0) {
      j = i / factor;
      sum[j] = std::numeric_limits<T>::max();
      answer[j] = j * factor;
    }
  }

  hipMalloc((void**)&ad, arraysize * factor * sizeof(T));
  hipMalloc((void**)&sumd, arraysize * sizeof(T));

  hipMemcpy(ad, a.data(), arraysize * factor * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(sumd, sum.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);

  min_test_kernel<<<dimGrid, dimBlock>>>(ad, sumd);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipMemcpy(sum.data(), sumd, arraysize * sizeof(T), hipMemcpyDeviceToHost);

  for (int i = 0; i < arraysize; ++i) {
    ASSERT_EQ(sum[i], answer[i]) << typeid(T).name();
  }

  hipFree(ad);
  hipFree(sumd);
}

TEST(TestAtomicOps, TestAtomicAdd) {
  if (!at::cuda::is_available()) return;
  test_atomic_add<uint8_t>();
  test_atomic_add<int8_t>();
  test_atomic_add<int16_t>();
  test_atomic_add<int32_t>();
  test_atomic_add<int64_t>();

  test_atomic_add<at::BFloat16>();
  test_atomic_add<at::Half>();
  test_atomic_add<float>();
  test_atomic_add<double>();
  test_atomic_add<c10::complex<float> >();
  test_atomic_add<c10::complex<double> >();
}

TEST(TestAtomicOps, DISABLED_ON_WINDOWS(TestAtomicMul)) {
  if (!at::cuda::is_available()) return;
  test_atomic_mul<uint8_t>();
  test_atomic_mul<int8_t>();
  test_atomic_mul<int16_t>();
  test_atomic_mul<int32_t>();
  test_atomic_mul<int64_t>();
  test_atomic_mul<at::BFloat16>();
  test_atomic_mul<at::Half>();
  test_atomic_mul<float>();
  test_atomic_mul<double>();
}

TEST(TestAtomicOps, DISABLED_ON_WINDOWS(TestAtomicMax)) {
  if (!at::cuda::is_available()) return;
  test_atomic_max<uint8_t>();
  test_atomic_max<int8_t>();
  test_atomic_max<int16_t>();
  test_atomic_max<int32_t>();
  test_atomic_max<int64_t>();
  test_atomic_max<at::BFloat16>();
  test_atomic_max<at::Half>();
  test_atomic_max<float>();
  test_atomic_max<double>();
}

TEST(TestAtomicOps, DISABLED_ON_WINDOWS(TestAtomicMin)) {
  if (!at::cuda::is_available()) return;
  test_atomic_min<uint8_t>();
  test_atomic_min<int8_t>();
  test_atomic_min<int16_t>();
  test_atomic_min<int32_t>();
  test_atomic_min<int64_t>();
  test_atomic_min<at::BFloat16>();
  test_atomic_min<at::Half>();
  test_atomic_min<float>();
  test_atomic_min<double>();
}
