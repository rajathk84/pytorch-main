#include <hip/hip_runtime.h>


__device__ void add(const float* a, const float* b, float* output) {
    *output = *a + *b;
}
