#include "hip/hip_runtime.h"
#include "CUDATest.hpp"
#include <ATen/cuda/Exceptions.h>

namespace c10d {
namespace test {

namespace {
__global__ void waitClocks(const uint64_t count) {
  // Few AMD specific GPUs have different clock intrinsic
#if defined(__GFX11__) && defined(USE_ROCM) && !defined(__CUDA_ARCH__)
  clock_t start = wall_clock64();
#else
  clock_t start = clock64();
#endif
  clock_t offset = 0;
  while (offset < count) {
    offset = clock() - start;
  }
}

} // namespace

void cudaSleep(at::cuda::CUDAStream& stream, uint64_t clocks) {
  waitClocks<<<1, 1, 0, stream.stream()>>>(clocks);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

int cudaNumDevices() {
  int n = 0;
  C10_CUDA_CHECK_WARN(hipGetDeviceCount(&n));
  return n;
}

} // namespace test
} // namespace c10d
