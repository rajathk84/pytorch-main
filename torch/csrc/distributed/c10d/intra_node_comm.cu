#include "hip/hip_runtime.h"
#include <torch/csrc/distributed/c10d/intra_node_comm.hpp>

#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

namespace c10d {
namespace intra_node_comm {

static constexpr size_t kBytesPerThread = 16;
static constexpr size_t kMaxAllReduceBlocks = 24;
static constexpr size_t kThreadsPerBlock = 1024;
static constexpr size_t kWarpSize = 32;

static constexpr size_t kHcmThreshBytes = 256 * 1024;
static constexpr size_t kOneShotThreshBytes = 256 * 1024;
static constexpr size_t kTwoShotThreshBytes = 10 * 1024 * 1024;

#if defined(USE_ROCM)
using __hip_bfloat162 = uint32_t;
#endif

struct __align__(16) bf16x8 {
  __hip_bfloat162 vals[4];
};

#define DEVICE_INLINE __device__ inline __attribute__((always_inline))

DEVICE_INLINE __hip_bfloat162
bf16hadd2(const __hip_bfloat162 x, const __hip_bfloat162 y) {
#if defined(USE_ROCM)
  CUDA_KERNEL_ASSERT(false);
  return 0;
#elif (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
  __hip_bfloat162 res;
  return res;
#else
  return __hadd2(x, y);
#endif
}

DEVICE_INLINE bf16x8 add_bf16x8(bf16x8 a, bf16x8 b) {
  bf16x8 c;
  c.vals[0] = bf16hadd2(a.vals[0], b.vals[0]);
  c.vals[1] = bf16hadd2(a.vals[1], b.vals[1]);
  c.vals[2] = bf16hadd2(a.vals[2], b.vals[2]);
  c.vals[3] = bf16hadd2(a.vals[3], b.vals[3]);
  return c;
}

/**
 * NOTE [cross device memory synchronization]
 *
 * The multi-stage algorithms (e.g. two-shot, hcm allreduce) require the writes
 * of a thread to be visible by threads with the same block/thread ID on other
 * devices. To satisfy CUDA's memory consistency model, every thread has to
 * release its writes at the system scope, and the consuming thread has to
 * acquire the writes at the system scope. This incurs high overhead and
 * attempts in optmizing this process can be prone to race condition.
 *
 * Instead, we go around caching by having each thread:
 *
 * - Directly write to global memory via st.cs (cache-streaming).
 * - Synchronize with threads within the block.
 * - Perform cross device synchronization at block level (via system scope
 *   atomic ops).
 * - Synchronize with threads within the block.
 * - Directly read from global memory via ld.nc (non-coherent/non-cached).
 */
template <typename T>
DEVICE_INLINE void streamLoad128(bf16x8& val, const T* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  asm("ld.global.nc.v2.u64 {%0, %1}, [%2];"
      : "=l"(low), "=l"(high)
      : "l"(addr));
  reinterpret_cast<unsigned long long int*>(&val)[0] = low;
  reinterpret_cast<unsigned long long int*>(&val)[1] = high;
#endif
}

__device__ inline void streamStore128(at::BFloat16* addr, const bf16x8& val) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  low = reinterpret_cast<const unsigned long long int*>(&val)[0];
  high = reinterpret_cast<const unsigned long long int*>(&val)[1];
  asm("st.global.cs.v2.u64 [%0], {%1, %2};" : : "l"(addr), "l"(low), "l"(high));
#endif
}

template <typename T>
DEVICE_INLINE void load128(bf16x8& val, const T* addr) {
  *reinterpret_cast<uint4*>(&val) = reinterpret_cast<const uint4*>(addr)[0];
}

template <typename T>
DEVICE_INLINE void store128(T* addr, const bf16x8& val) {
  *reinterpret_cast<uint4*>(addr) = reinterpret_cast<const uint4*>(&val)[0];
}

DEVICE_INLINE void releaseSignal(uint32_t* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  atomicAdd_system(addr, 1);
#endif
}

DEVICE_INLINE void acquireSignal(uint32_t* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  volatile uint32_t* signal = addr;
  uint32_t val;
  do {
    val = *signal;
  } while (val == 0 || atomicCAS_system(addr, val, val - 1) != val);
#endif
}

////////////////////////////////////////////////////////////////////////////////
// Fully Connected Algos
////////////////////////////////////////////////////////////////////////////////

struct P2pState {
  uint32_t signals0[kMaxAllReduceBlocks][kMaxDevices];
  uint32_t signals1[kMaxAllReduceBlocks][kMaxDevices];
};

static_assert(sizeof(P2pState) <= kP2pStateSize);

template <uint32_t kWorldSize, bool kAligned>
static __global__ void oneShotAllReduceKernel(
    at::BFloat16* input,
    size_t N,
    size_t N_aligned,
    P2pState** p2pStates,
    at::BFloat16** buffers,
    size_t rank,
    bool fuseInputCopy) {
  const size_t numelPerThread = kBytesPerThread / sizeof(at::BFloat16);
  const size_t offset =
      (blockDim.x * blockIdx.x + threadIdx.x) * numelPerThread;
  const size_t stride = blockDim.x * gridDim.x * numelPerThread;

  if (fuseInputCopy) {
    for (size_t i = offset; i < N_aligned; i += stride) {
      bf16x8 val;
      streamLoad128(val, &input[i]);
      streamStore128(&buffers[rank][i], val);
    }
  }

  // Wait for all other ranks to enter the kernel
  if (threadIdx.x < kWorldSize) {
    auto targetRank = threadIdx.x;
    releaseSignal(&p2pStates[targetRank]->signals0[blockIdx.x][rank]);
    acquireSignal(&p2pStates[rank]->signals0[blockIdx.x][targetRank]);
  }
  __syncthreads();

  // The source pointers. Distributed round-robin for the different warps
  const at::BFloat16* srcs[kWorldSize];
#pragma unroll kWorldSize
  for (int ii = 0; ii < kWorldSize; ++ii) {
    int srcRank = (rank + ii) % kWorldSize;
    srcs[ii] = buffers[srcRank];
  }

  for (size_t i = offset; i < N_aligned; i += stride) {
    bf16x8 vals[kWorldSize];
#pragma unroll kWorldSize
    for (size_t ii = 0; ii < kWorldSize; ++ii) {
      // Make sure the values in `vals` are order by rank so that the reduction
      // results are consistent across ranks.
      int srcRank = (ii + kWorldSize - rank) % kWorldSize;
      streamLoad128(vals[srcRank], &srcs[ii][i]);
    }

    bf16x8 sums;
    memset(reinterpret_cast<void*>(&sums), 0, sizeof(sums));

#pragma unroll kWorldSize
    for (size_t ii = 0; ii < kWorldSize; ++ii) {
      sums = add_bf16x8(sums, vals[ii]);
    }
    if constexpr (kAligned) {
      streamStore128(&input[i], sums);
    } else {
      for (size_t ii = 0; ii < numelPerThread; ++ii) {
        if (i + ii < N) {
          input[i + ii] = reinterpret_cast<at::BFloat16*>(&sums)[ii];
        }
      }
    }
  }
}

template <uint32_t kWorldSize>
static __launch_bounds__(1024) __global__ void twoShotAllReduceKernel(
    at::BFloat16* input,
    size_t N_aligned,
    P2pState** p2pStates,
    at::BFloat16** buffers,
    size_t rank) {
  const size_t numelPerThread = kBytesPerThread / sizeof(at::BFloat16);
  const size_t offset =
      (blockDim.x * blockIdx.x + threadIdx.x) * numelPerThread;
  const size_t stride = blockDim.x * gridDim.x * numelPerThread;
  const size_t N_per_rank = N_aligned / kWorldSize;
  const size_t N_start = N_per_rank * rank;

  // Wait for all other ranks to enter the kernel
  if (threadIdx.x < kWorldSize) {
    auto targetRank = threadIdx.x;
    releaseSignal(&p2pStates[targetRank]->signals0[blockIdx.x][rank]);
    acquireSignal(&p2pStates[rank]->signals0[blockIdx.x][targetRank]);
  }
  __syncthreads();

  // The source pointers. Distributed round-robin for the different warps
  at::BFloat16* srcs[kWorldSize];
  size_t srcRanks[kWorldSize];
#pragma unroll kWorldSize
  for (int ii = 0; ii < kWorldSize; ++ii) {
    int srcRank = (rank + ii) % kWorldSize;
    srcs[ii] = buffers[srcRank];
    srcRanks[ii] = srcRank;
  }

  for (size_t i = offset; i < N_per_rank; i += stride) {
    bf16x8 vals[kWorldSize];
#pragma unroll kWorldSize
    for (size_t ii = 0; ii < kWorldSize; ++ii) {
      // Make sure the values in `vals` are order by rank so that the reduction
      // results are consistent across ranks.
      int srcRank = (ii + kWorldSize - rank) % kWorldSize;
      streamLoad128(vals[srcRank], &srcs[ii][N_start + i]);
    }

    bf16x8 sums;
    memset(reinterpret_cast<void*>(&sums), 0, sizeof(sums));

#pragma unroll kWorldSize
    for (size_t ii = 0; ii < kWorldSize; ++ii) {
      sums = add_bf16x8(sums, vals[ii]);
    }
    streamStore128(&srcs[0][N_start + i], sums);
    // Store local sums into input now so we can avoid
    // a global memory access later for it.
    streamStore128(&input[N_start + i], sums);
  }
  __syncthreads();

  if (threadIdx.x < kWorldSize) {
    auto targetRank = threadIdx.x;
    releaseSignal(&p2pStates[targetRank]->signals1[blockIdx.x][rank]);
    acquireSignal(&p2pStates[rank]->signals1[blockIdx.x][targetRank]);
  }
  __syncthreads();

  for (size_t i = offset; i < N_per_rank; i += stride) {
#pragma unroll kWorldSize - 1
    for (size_t ii = 1; ii < kWorldSize; ++ii) {
      size_t k = N_start + i + (srcRanks[ii] - rank) * N_per_rank;
      bf16x8 val;
      streamLoad128(val, &srcs[ii][k]);
      streamStore128(&input[k], val);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Hybrid Cube Mesh Algos
////////////////////////////////////////////////////////////////////////////////

/**
 * NOTE [hybrid cube mesh]
 *
 * In a hybrid cube mesh topology, every device has exactly 4 neighbors
 * (directly connected via NVLink). For every device X, it has exactly 1
 * neighbor Y that is a neighbor of the 3 non-neighbor of X. We call Y the
 * relay neighbor of X. This property is symmetrical: X is also guaranteed to
 * be the relay neighbor of Y.
 *
 * With this property, we can perform a variant of one-shot allreduce algo that
 * only moves data across NVLinks:
 *
 * - Each device one-shot allreduce among itself and 3 non-relay neighbors.
 * - Each device exchange data with its relay neighbor.
 *
 * HybridCubeMesh is a data structure for describing the topology:
 *
 * - hcm[X][0:3] are the 3 neighbors of X.
 * - hcm[X][3] is the relay neighbor of X.
 * - For load balancing purpose, we also ensure that if hcm[X][k] = Y,
 *   hcm[Y][k] = X.
 */
std::optional<HybridCubeMesh> getHybridCubeMesh(NvlMesh nvlMesh) {
  std::array<std::unordered_set<size_t>, kMaxDevices> neighbors = {};
  std::array<size_t, kMaxDevices> neighborMasks = {};
  for (size_t i = 0; i < kMaxDevices; ++i) {
    for (size_t j = 0; j < kMaxDevices; ++j) {
      if (nvlMesh[i][j] > 0) {
        neighbors[i].insert(j);
        neighborMasks[i] |= (1ul << j);
      }
    }
  }
  HybridCubeMesh hcm = {};
  for (auto& row : hcm) {
    row.fill(-1);
  }
  // A topology is an HCM if:
  // - Every device has exactly 4 neighbors.
  // - For every device, it has exactly 1 relay neighbor that is
  //   a neighbor of the 3 non-neighbor of the device.
  for (size_t i = 0; i < kMaxDevices; ++i) {
    if (neighbors[i].size() != 4) {
      return std::nullopt;
    }
    // Condition 1: check the number of neighbors
    std::vector<size_t> relayNeighbors;
    for (size_t j = 0; j < kMaxDevices; ++j) {
      if ((neighborMasks[i] & neighborMasks[j]) == 0) {
        relayNeighbors.push_back(j);
      }
    }
    // Condition 2: check the number of relay neighbors
    if (relayNeighbors.size() != 1) {
      return std::nullopt;
    }
    neighbors[i].erase(relayNeighbors[0]);
    hcm[i][3] = relayNeighbors[0];
  }

  for (size_t i = 0; i < kMaxDevices; ++i) {
    for (size_t k = 0; k < 3; ++k) {
      // We can only fill hcm[i][k] with j if hcm[j][k] is not filled
      for (size_t j : neighbors[i]) {
        if (hcm[j][k] == -1) {
          hcm[i][k] = j;
          hcm[j][k] = i;
          break;
        }
      }
      TORCH_CHECK(hcm[i][k] != -1);
      neighbors[i].erase(hcm[i][k]);
    }
  }
  return hcm;
}

template <bool kAligned>
static __global__ void hybridCubeMeshAllReduceKernel(
    at::BFloat16* input,
    size_t N,
    size_t N_aligned,
    P2pState** p2pStates,
    at::BFloat16** buffers,
    int hcmInfo[4],
    size_t bufferSize,
    size_t rank) {
  const size_t numelPerThread = kBytesPerThread / sizeof(at::BFloat16);
  const size_t offset =
      (blockDim.x * blockIdx.x + threadIdx.x) * numelPerThread;
  const size_t stride = blockDim.x * gridDim.x * numelPerThread;
  const int relayRank = hcmInfo[3];

  // Wait for HCM neigbors to enter the kernel
  if (threadIdx.x < 3) {
    auto targetRank = hcmInfo[threadIdx.x];
    releaseSignal(&p2pStates[targetRank]->signals0[blockIdx.x][rank]);
    acquireSignal(&p2pStates[rank]->signals0[blockIdx.x][targetRank]);
  }
  __syncthreads();

  const at::BFloat16* srcs[4] = {
      buffers[rank],
      buffers[hcmInfo[0]],
      buffers[hcmInfo[1]],
      buffers[hcmInfo[2]],
  };
  // Use the half second half of the buffer as relay
  at::BFloat16* localRelay =
      buffers[rank] + (bufferSize / sizeof(at::BFloat16) / 2);
  at::BFloat16* remoteRelay =
      buffers[relayRank] + (bufferSize / sizeof(at::BFloat16) / 2);

  for (size_t i = offset; i < N_aligned; i += stride) {
    bf16x8 vals[4];

#pragma unroll 4
    for (size_t ii = 0; ii < 4; ++ii) {
      streamLoad128(vals[ii], &srcs[ii][i]);
    }

    bf16x8 sums;
    memset(reinterpret_cast<void*>(&sums), 0, sizeof(sums));

#pragma unroll 4
    for (size_t ii = 0; ii < 4; ++ii) {
      sums = add_bf16x8(sums, vals[ii]);
    }
    // Cached store for local sums
    store128(&localRelay[i], sums);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    releaseSignal(&p2pStates[relayRank]->signals0[blockIdx.x][rank]);
    acquireSignal(&p2pStates[rank]->signals0[blockIdx.x][relayRank]);
  }
  __syncthreads();

  for (size_t i = offset; i < N_aligned; i += stride) {
    bf16x8 localSum, remoteSum;
    // Cached load for local sums
    load128(localSum, &localRelay[i]);
    streamLoad128(remoteSum, &remoteRelay[i]);
    localSum = add_bf16x8(localSum, remoteSum);
    if constexpr (kAligned) {
      streamStore128(&input[i], localSum);
    } else {
      for (size_t ii = 0; ii < numelPerThread; ++ii) {
        if (i + ii < N) {
          input[i + ii] = reinterpret_cast<at::BFloat16*>(&localSum)[ii];
        }
      }
    }
  }
}

static inline size_t divUp(uint32_t a, uint32_t b) {
  return (a + b - 1) / b;
}

static inline size_t alignUp(uint32_t a, uint32_t b) {
  return divUp(a, b) * b;
}

static void checkInput(const at::Tensor& input, int deviceIdx) {
  TORCH_CHECK(
      input.dtype() == at::kBFloat16,
      "oneShotAllReduce only supports bf16 for now");
  TORCH_CHECK(input.is_non_overlapping_and_dense());
  TORCH_CHECK(input.device().is_cuda());
  TORCH_CHECK(
      input.get_device() == deviceIdx,
      "IntraNodeComm: expect input to be on device ",
      deviceIdx,
      ", got device ",
      input.get_device());
}

static void getLaunchConfig(
    size_t N_aligned,
    size_t elemSize,
    dim3& blocks,
    dim3& threads) {
  blocks = dim3(0, 1, 1);
  threads = dim3(0, 1, 1);

  const auto numelPerThread = kBytesPerThread / elemSize;
  const auto numelPerWarp = numelPerThread * kWarpSize;
  TORCH_CHECK(N_aligned % numelPerThread == 0);
  TORCH_CHECK(N_aligned % numelPerWarp == 0);
  if (N_aligned < numelPerThread * kThreadsPerBlock) {
    threads.x = N_aligned / numelPerWarp * kWarpSize;
    blocks.x = 1;
  } else {
    auto warpsRequired = N_aligned / numelPerWarp;
    auto threadsRequired = N_aligned / numelPerThread;
    blocks.x =
        std::min(divUp(threadsRequired, kThreadsPerBlock), kMaxAllReduceBlocks);
    auto warpsPerBlock = divUp(warpsRequired, blocks.x);
    threads.x = std::min(kThreadsPerBlock, warpsPerBlock * kWarpSize);
  }
}

bool isIntraNodeCommSupported() {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  return false;
#else
  return true;
#endif
}

void* initP2pState() {
  void* state = nullptr;
  AT_CUDA_CHECK(hipMalloc(&state, sizeof(P2pState)));
  AT_CUDA_CHECK(hipMemset(state, 0, sizeof(P2pState)));
  return state;
}

void* initTopoInfo(Topology topology, NvlMesh nvlMesh, size_t rank) {
  void* topoInfo = nullptr;
  if (topology != Topology::HYBRID_CUBE_MESH) {
    return topoInfo;
  }
  auto hcm = getHybridCubeMesh(nvlMesh);
  int hcmInfo[4];
  std::copy((*hcm)[rank].begin(), (*hcm)[rank].begin() + 4, hcmInfo);
  AT_CUDA_CHECK(hipMalloc(&topoInfo, sizeof(hcmInfo)));
  AT_CUDA_CHECK(
      hipMemcpy(topoInfo, hcmInfo, sizeof(hcmInfo), hipMemcpyHostToDevice));
  return topoInfo;
}

at::Tensor IntraNodeComm::oneShotAllReduce(
    const at::Tensor& input,
    at::cuda::CUDAStream& stream) {
  checkInput(input, deviceIdx_);

  const size_t numelPerWarp =
      kBytesPerThread / input.element_size() * kWarpSize;
  const size_t N_aligned = alignUp(input.numel(), numelPerWarp);
  const bool isAligned = (N_aligned == static_cast<size_t>(input.numel()));
  TORCH_CHECK(N_aligned <= bufferSize_ / input.element_size());

  dim3 blocks, threads;
  getLaunchConfig(N_aligned, input.element_size(), blocks, threads);

  at::cuda::OptionalCUDAGuard guard(input.get_device());

  // When the input data is small, copying inside the kernel is faster. Because
  // in such cases, the launch overhead of hipMemcpyAsync outweighs its
  // efficiency. Here we consider the input data to be small if the copy loop
  // can finish in a single iteration.
  const bool fuseInputCopy = isAligned && blocks.x < kMaxAllReduceBlocks;
  if (!fuseInputCopy) {
    AT_CUDA_CHECK(hipMemcpyAsync(
        symmetricMemory_->get_buffer_ptrs()[rank_],
        input.data_ptr(),
        input.numel() * input.element_size(),
        hipMemcpyDeviceToDevice,
        stream));
  }

#define X(kWorldSize, kAligned)                            \
  if (worldSize_ == kWorldSize) {                          \
    oneShotAllReduceKernel<kWorldSize, kAligned>           \
        <<<blocks, threads, 0, stream>>>(                  \
            input.data_ptr<at::BFloat16>(),                \
            input.numel(),                                 \
            N_aligned,                                     \
            reinterpret_cast<P2pState**>(p2pStatesDev_),   \
            reinterpret_cast<at::BFloat16**>(buffersDev_), \
            rank_,                                         \
            fuseInputCopy);                                \
    C10_CUDA_KERNEL_LAUNCH_CHECK();                        \
  }

#define DISPATCH_ALL_WORLD_SIZES(kAligned) \
  X(2, kAligned);                          \
  X(3, kAligned);                          \
  X(4, kAligned);                          \
  X(5, kAligned);                          \
  X(6, kAligned);                          \
  X(7, kAligned);                          \
  X(8, kAligned);

  if (isAligned) {
    DISPATCH_ALL_WORLD_SIZES(true);
  } else {
    DISPATCH_ALL_WORLD_SIZES(false);
  }

#undef DISPATCH_ALL_WORLD_SIZES
#undef X
  return input;
}

at::Tensor IntraNodeComm::twoShotAllReduce(
    const at::Tensor& input,
    at::cuda::CUDAStream& stream) {
  checkInput(input, deviceIdx_);

  size_t numelPerWarp = kBytesPerThread / input.element_size() * kWarpSize;
  size_t N_aligned = alignUp(input.numel(), worldSize_ * numelPerWarp);
  size_t N_per_rank = N_aligned / worldSize_;
  TORCH_CHECK(N_aligned <= bufferSize_ / input.element_size());

  dim3 blocks, threads;
  getLaunchConfig(N_per_rank, input.element_size(), blocks, threads);

  auto output = N_aligned == static_cast<size_t>(input.numel())
      ? input
      : input.new_empty(N_aligned);

  at::cuda::OptionalCUDAGuard guard(input.get_device());
  AT_CUDA_CHECK(hipMemcpyAsync(
      symmetricMemory_->get_buffer_ptrs()[rank_],
      input.data_ptr(),
      input.numel() * input.element_size(),
      hipMemcpyDeviceToDevice,
      stream));

#define X(kWorldSize)                                                   \
  if (worldSize_ == kWorldSize) {                                       \
    twoShotAllReduceKernel<kWorldSize><<<blocks, threads, 0, stream>>>( \
        output.data_ptr<at::BFloat16>(),                                \
        N_aligned,                                                      \
        reinterpret_cast<P2pState**>(p2pStatesDev_),                    \
        reinterpret_cast<at::BFloat16**>(buffersDev_),                  \
        rank_);                                                         \
    C10_CUDA_KERNEL_LAUNCH_CHECK();                                     \
  }
  X(2);
  X(3);
  X(4);
  X(5);
  X(6);
  X(7);
  X(8);
#undef X

  if (output.data_ptr() != input.data_ptr()) {
    AT_CUDA_CHECK(hipMemcpyAsync(
        input.data_ptr(),
        output.data_ptr(),
        input.numel() * input.element_size(),
        hipMemcpyDeviceToDevice,
        stream));
  }
  return input;
}

at::Tensor IntraNodeComm::hybridCubeMeshAllReduce(
    const at::Tensor& input,
    at::cuda::CUDAStream& stream) {
  checkInput(input, deviceIdx_);

  size_t numelPerWarp = kBytesPerThread / input.element_size() * kWarpSize;
  size_t N_aligned = alignUp(input.numel(), numelPerWarp);
  TORCH_CHECK(N_aligned * 2 <= bufferSize_ / input.element_size());

  dim3 blocks, threads;
  getLaunchConfig(N_aligned, input.element_size(), blocks, threads);

  at::cuda::OptionalCUDAGuard guard(input.get_device());
  AT_CUDA_CHECK(hipMemcpyAsync(
      symmetricMemory_->get_buffer_ptrs()[rank_],
      input.data_ptr(),
      input.numel() * input.element_size(),
      hipMemcpyDeviceToDevice,
      stream));

#define X(kAligned)                                                        \
  hybridCubeMeshAllReduceKernel<kAligned><<<blocks, threads, 0, stream>>>( \
      input.data_ptr<at::BFloat16>(),                                      \
      input.numel(),                                                       \
      N_aligned,                                                           \
      reinterpret_cast<P2pState**>(p2pStatesDev_),                         \
      reinterpret_cast<at::BFloat16**>(buffersDev_),                       \
      static_cast<int*>(topoInfo_),                                        \
      bufferSize_,                                                         \
      rank_);                                                              \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  if (N_aligned == static_cast<size_t>(input.numel())) {
    X(true);
  } else {
    X(false);
  }
#undef X
  return input;
}

AllReduceAlgo IntraNodeComm::selectAllReduceAlgo(const at::Tensor& input) {
  // Only support bf16 for now
  if (input.dtype() != at::kBFloat16) {
    return AllReduceAlgo::NONE;
  }
  const auto inputSize = input.numel() * input.element_size();
  const auto bytesPerWarp = kBytesPerThread * kWarpSize;

  if (topology_ == Topology::HYBRID_CUBE_MESH) {
    TORCH_CHECK(
        worldSize_ == 8, "hyperCubeAllReduce only supports exactly 8 GPUs");
    const auto hcmInputSize = alignUp(inputSize, bytesPerWarp);
    const auto hcmBufferSizeReq = hcmInputSize * 2;
    if (hcmInputSize <= kHcmThreshBytes && hcmBufferSizeReq <= bufferSize_) {
      return AllReduceAlgo::HCM;
    }
  }
  if (topology_ == Topology::FULLY_CONNECTED) {
    const auto oneShotInputSize = alignUp(inputSize, bytesPerWarp);
    const auto oneShotBufferSizeReq = oneShotInputSize;
    if (oneShotInputSize <= kOneShotThreshBytes &&
        oneShotBufferSizeReq <= bufferSize_) {
      return AllReduceAlgo::ONE_SHOT;
    }

    const auto twoShotInputSize = alignUp(inputSize, bytesPerWarp * worldSize_);
    const auto twoShotBufferSizeReq = twoShotInputSize;
    if (twoShotInputSize <= kTwoShotThreshBytes &&
        twoShotBufferSizeReq <= bufferSize_) {
      return AllReduceAlgo::TWO_SHOT;
    }
  }
  return AllReduceAlgo::NONE;
}

static int64_t usageCounter = 0;

at::Tensor IntraNodeComm::allReduce(
    const at::Tensor& input,
    AllReduceAlgo algo) {
  // Report usage for testing purposes.
  // We don't care about overflowing.
  ++usageCounter;
  auto stream = at::cuda::getCurrentCUDAStream();
  c10::cuda::CUDACachingAllocator::recordStream(
      input.storage().data_ptr(), stream);
  switch (algo) {
    case AllReduceAlgo::ONE_SHOT:
      return oneShotAllReduce(input, stream);
    case AllReduceAlgo::TWO_SHOT:
      return twoShotAllReduce(input, stream);
    case AllReduceAlgo::HCM:
      return hybridCubeMeshAllReduce(input, stream);
    default:
      C10_THROW_ERROR(ValueError, "IntraNodeComm: invalid algo");
  }
}

int64_t getIntraNodeCommUsageCounter() {
  return usageCounter;
}

static __global__ void barrierKernel(
    P2pState** p2pStates,
    uint64_t mask,
    size_t rank,
    size_t worldSize) {
  if (threadIdx.x < worldSize && (mask & (1ULL << threadIdx.x))) {
    auto targetRank = threadIdx.x;
    releaseSignal(&p2pStates[targetRank]->signals0[0][rank]);
    acquireSignal(&p2pStates[rank]->signals0[0][targetRank]);
  }
}

void IntraNodeComm::barrier(std::optional<std::vector<int64_t>> ranks) {
  barrierReady_.block(at::cuda::getCurrentCUDAStream());
  if (!ranks.has_value()) {
    ranks = std::vector<int64_t>(worldSize_);
    std::iota(ranks->begin(), ranks->end(), 0);
  }
  uint64_t mask = 0;
  for (const auto& r : ranks.value()) {
    TORCH_CHECK(r >= 0 && r < static_cast<int64_t>(worldSize_));
    mask |= (1ULL << r);
  }
  barrierKernel<<<1, kWarpSize, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<P2pState**>(p2pStatesDev_), mask, rank_, worldSize_);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  barrierReady_.record();
}

at::Tensor IntraNodeComm::getBuffer(
    size_t rank,
    const std::vector<int64_t>& sizes,
    c10::ScalarType dtype,
    int64_t storageOffset) {
  return symmetricMemory_->get_buffer(rank, sizes, dtype, storageOffset);
}

} // namespace intra_node_comm
} // namespace c10d
