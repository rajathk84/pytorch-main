#include "hip/hip_runtime.h"
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory.hpp>

#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAGuard.h>

#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
#include <c10/cuda/driver_api.h>
#endif

#include <sys/socket.h>
#include <sys/syscall.h>
#include <sys/un.h>
#include <unistd.h>

#if defined(CUDART_VERSION) && CUDART_VERSION >= 12030
#define CUDART_SUPPORTS_MULTICAST
#endif

namespace {

bool has_multicast_support() {
#if defined(CUDART_SUPPORTS_MULTICAST)
  return c10::cuda::DriverAPI::get()->cuMulticastCreate_ != nullptr;
#else
  return false;
#endif
}

class IpcChannel {
 public:
  IpcChannel() : socket_name_(get_socket_name(getpid())) {
    TORCH_CHECK(
        (socket_ = socket(AF_UNIX, SOCK_DGRAM, 0)) != 0,
        "Failed to create socket: ",
        strerror(errno));

    struct sockaddr_un addr = {.sun_family = AF_UNIX};
    std::copy(socket_name_.begin(), socket_name_.end(), addr.sun_path);

    TORCH_CHECK(
        bind(socket_, (struct sockaddr*)&addr, SUN_LEN(&addr)) == 0,
        "Failed to bind socket: ",
        strerror(errno));
  }

  ~IpcChannel() {
    close(socket_);
    unlink(socket_name_.c_str());
  }

  void send_fd(int dst_pid, int fd) {
    struct sockaddr_un addr = {.sun_family = AF_UNIX};
    auto socket_name = get_socket_name(dst_pid);
    std::copy(socket_name.begin(), socket_name.end(), addr.sun_path);

    struct iovec io = {.iov_base = (void*)("fd"), .iov_len = 2};

    char cbuf[CMSG_SPACE(sizeof(int))];
    memset(cbuf, 0, sizeof(cbuf));

    struct msghdr msg {
      .msg_name = (void*)&addr, .msg_namelen = sizeof(struct sockaddr_un),
      .msg_iov = &io, .msg_iovlen = 1, .msg_control = cbuf,
      .msg_controllen = sizeof(cbuf)
    };

    auto cmsg = CMSG_FIRSTHDR(&msg);
    cmsg->cmsg_len = CMSG_LEN(sizeof(int));
    cmsg->cmsg_level = SOL_SOCKET;
    cmsg->cmsg_type = SCM_RIGHTS;
    memcpy(CMSG_DATA(cmsg), &fd, sizeof(fd));

    TORCH_CHECK(
        sendmsg(socket_, &msg, 0) > 0, "Failed to send fd: ", strerror(errno));
  }

  int recv_fd() {
    char buf[2];
    struct iovec io = {.iov_base = (void*)buf, .iov_len = sizeof(buf)};

    char cbuf[CMSG_SPACE(sizeof(int))];
    memset(cbuf, 0, sizeof(cbuf));

    struct msghdr msg = {
        .msg_iov = &io,
        .msg_iovlen = 1,
        .msg_control = cbuf,
        .msg_controllen = sizeof(cbuf)};

    TORCH_CHECK(
        recvmsg(socket_, &msg, 0) > 0,
        "Failed to receive fd: ",
        strerror(errno));

    auto cmsg = CMSG_FIRSTHDR(&msg);
    TORCH_CHECK(cmsg != NULL);
    TORCH_CHECK(cmsg->cmsg_len == CMSG_LEN(sizeof(int)));
    TORCH_CHECK(
        cmsg->cmsg_level == SOL_SOCKET && cmsg->cmsg_type == SCM_RIGHTS);
    return *reinterpret_cast<int*>(CMSG_DATA(cmsg));
  }

  std::vector<int> all_gather_fds(
      int rank,
      const std::vector<int>& pids,
      int fd) {
    size_t world_size = pids.size();
    std::vector<int> fds(pids.size());
    fds[rank] = fd;

    int dst_rank = (rank + 1) % world_size;
    for (size_t step = 1; step < world_size; ++step) {
      int src_rank = (rank + world_size - step) % world_size;
      send_fd(pids[dst_rank], fd);
      fd = recv_fd();
      fds[src_rank] = fd;
    }
    return fds;
  }

  int broadcast_fds(
      int rank,
      int src_rank,
      const std::vector<int>& pids,
      int fd) {
    size_t world_size = pids.size();

    if (rank == src_rank) {
      for (int dst_rank = 0; dst_rank < (int)world_size; ++dst_rank) {
        if (dst_rank == rank) {
          continue;
        }
        send_fd(pids[dst_rank], fd);
      }
      return fd;
    }
    return recv_fd();
  }

 private:
  static std::string get_socket_name(int pid) {
    const char* tmp_dir = "/tmp";
    for (const char* env_var : {"TMPDIR", "TMP", "TEMP", "TEMPDIR"}) {
      if (const char* path = getenv(env_var)) {
        tmp_dir = path;
        break;
      }
    }
    std::ostringstream oss;
    oss << tmp_dir << "/symm_mem-" << pid;
    return oss.str();
  }

  std::string socket_name_;
  int socket_;
};

constexpr size_t signal_pad_size = 2048;
const std::string store_comm_prefix = "CUDASymmetricMemory";

static size_t store_comm_seq_id = 0;

template <typename T>
std::vector<T> store_all_gather(
    const c10::intrusive_ptr<c10d::Store>& store,
    int rank,
    int world_size,
    T val) {
  static_assert(std::is_trivially_copyable_v<T>);

  std::vector<std::string> peer_keys;
  for (int r = 0; r < world_size; ++r) {
    std::ostringstream oss;
    oss << store_comm_prefix << "/" << store_comm_seq_id << "/" << r;
    peer_keys.push_back(oss.str());
  }
  ++store_comm_seq_id;

  {
    std::vector<uint8_t> payload(
        reinterpret_cast<uint8_t*>(&val),
        reinterpret_cast<uint8_t*>(&val) + sizeof(T));
    store->set(peer_keys[rank], payload);
  }

  std::vector<T> peer_vals;
  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      peer_vals.push_back(val);
      continue;
    }
    store->wait({peer_keys[r]});
    auto payload = store->get(peer_keys[r]);
    TORCH_CHECK(payload.size() == sizeof(T));
    T peer_val{};
    std::memcpy(&peer_val, payload.data(), sizeof(T));
    peer_vals.push_back(peer_val);
  }
  return peer_vals;
}

void store_barrier(
    const c10::intrusive_ptr<c10d::Store>& store,
    int rank,
    int world_size) {
  store_all_gather(store, rank, world_size, 0);
}

void map_block(
    void** ptr,
    c10d::symmetric_memory::HandleType handle,
    size_t size,
    int device_idx) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto driver_api = c10::cuda::DriverAPI::get();
  auto dev_ptr = reinterpret_cast<hipDeviceptr_t*>(ptr);
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMemAddressReserve_(dev_ptr, size, 0ULL, 0, 0ULL));
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemMap_(*dev_ptr, size, 0, handle, 0ULL));

  hipMemAccessDesc desc;
  desc.location.type = hipMemLocationTypeDevice;
  // NOLINTNEXTLINE(bugprone-signed-char-misuse)
  desc.location.id = static_cast<int>(device_idx);
  desc.flags = hipMemAccessFlagsProtReadWrite;
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemSetAccess_(*dev_ptr, size, &desc, 1));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

} // namespace

namespace c10d {
namespace symmetric_memory {

CUDASymmetricMemory::CUDASymmetricMemory(
    std::vector<HandleType> handles,
    size_t block_size,
    std::vector<void*> buffers,
    std::vector<void*> signal_pads,
    HandleType mc_handle,
    void* mc_addr,
    size_t buffer_size,
    int local_device_idx,
    int rank,
    int world_size)
    : handles_(std::move(handles)),
      block_size_(block_size),
      buffers_(std::move(buffers)),
      signal_pads_(std::move(signal_pads)),
      mc_handle_(mc_handle),
      mc_addr_(mc_addr),
      buffer_size_(buffer_size),
      local_device_idx_(local_device_idx),
      rank_(rank),
      world_size_(world_size) {
  const size_t arr_size = sizeof(void*) * world_size_;
  buffers_dev_ = reinterpret_cast<void**>(
      c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));
  signal_pads_dev_ = reinterpret_cast<void**>(
      c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));

  c10::cuda::CUDAGuard guard(local_device_idx);
  AT_CUDA_CHECK(hipMemcpy(
      buffers_dev_, buffers_.data(), arr_size, hipMemcpyHostToDevice));
  AT_CUDA_CHECK(hipMemcpy(
      signal_pads_dev_, signal_pads_.data(), arr_size, hipMemcpyHostToDevice));
}

CUDASymmetricMemory::~CUDASymmetricMemory() {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  // Leak the cuda allocations during static deinitialization
  if (is_finalizing()) {
    return;
  }
  c10::cuda::CUDAGuard guard(local_device_idx_);
  C10_CUDA_CHECK(hipDeviceSynchronize());

  auto driver_api = c10::cuda::DriverAPI::get();
  for (int r = 0; r < world_size_; ++r) {
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemUnmap_(
        reinterpret_cast<hipDeviceptr_t>(buffers_[r]), block_size_));
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemRelease_(handles_[r]));
  }
  c10::cuda::CUDACachingAllocator::raw_delete(buffers_dev_);
  c10::cuda::CUDACachingAllocator::raw_delete(signal_pads_dev_);
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

std::vector<void*> CUDASymmetricMemory::get_buffer_ptrs() {
  return buffers_;
}

std::vector<void*> CUDASymmetricMemory::get_signal_pad_ptrs() {
  return signal_pads_;
}

void** CUDASymmetricMemory::get_buffer_ptrs_dev() {
  return buffers_dev_;
}

void** CUDASymmetricMemory::get_signal_pad_ptrs_dev() {
  return signal_pads_dev_;
}

size_t CUDASymmetricMemory::get_buffer_size() {
  return buffer_size_;
}

size_t CUDASymmetricMemory::get_signal_pad_size() {
  return signal_pad_size;
}

bool CUDASymmetricMemory::has_multicast_support() {
  return ::has_multicast_support();
}

void* CUDASymmetricMemory::get_multicast_ptr() {
  return mc_addr_;
}

at::Tensor CUDASymmetricMemory::get_buffer(
    int rank,
    c10::IntArrayRef sizes,
    c10::ScalarType dtype,
    int64_t storage_offset) {
  const auto numel =
      std::accumulate(sizes.begin(), sizes.end(), 1, std::multiplies<int>());
  const auto element_size = c10::elementSize(dtype);
  const auto req_size = (numel + storage_offset) * element_size;
  TORCH_CHECK(
      req_size <= buffer_size_,
      "CUDASymmetricMemory::get_buffer: the requested size (",
      req_size,
      " bytes) exceeds the allocated size (",
      buffer_size_,
      " bytes)");
  auto device = c10::Device(c10::DeviceType::CUDA, local_device_idx_);
  auto options = at::TensorOptions().dtype(dtype).device(device);
  return at::for_blob(buffers_[rank], sizes)
      .storage_offset(storage_offset)
      .options(options)
      .target_device(device)
      .make_tensor();
}

void check_channel(int channel, int world_size) {
  TORCH_CHECK(
      channel >= 0,
      "channel for barrier(), put_signal() and wait_signal() ",
      "must be greater than 0 (got ",
      channel,
      ")");
  const size_t num_channels = signal_pad_size / sizeof(uint32_t) * world_size;
  TORCH_CHECK(
      static_cast<size_t>(channel) < num_channels,
      "The maximum supported channel for barrier(), put_signal() and wait_signal() is ",
      num_channels - 1,
      " (got ",
      channel,
      ")");
}

__device__ __forceinline__ void release_signal(uint32_t* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  volatile uint32_t* signal = addr;
  uint32_t val;
  do {
    val = *signal;
  } while (val != 0 || atomicCAS_system(addr, 0, 1) != 0);
#endif
}

__device__ __forceinline__ void acquire_signal(uint32_t* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  volatile uint32_t* signal = addr;
  uint32_t val;
  do {
    val = *signal;
  } while (val != 1 || atomicCAS_system(addr, 1, 0) != 1);
#endif
}

static __global__ void barrier_kernel(
    uint32_t** signal_pads,
    int channel,
    int rank,
    int world_size) {
  if (threadIdx.x < world_size) {
    auto target_rank = threadIdx.x;
    release_signal(signal_pads[target_rank] + world_size * channel + rank);
    acquire_signal(signal_pads[rank] + world_size * channel + target_rank);
  }
}

void CUDASymmetricMemory::barrier(int channel) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  barrier_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      channel,
      rank_,
      world_size_);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

static __global__ void put_signal_kernel(
    uint32_t** signal_pads,
    int dst_rank,
    int channel,
    int rank,
    int world_size) {
  if (threadIdx.x == 0) {
    release_signal(signal_pads[dst_rank] + world_size * channel + rank);
  }
}

void CUDASymmetricMemory::put_signal(int dst_rank, int channel) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  put_signal_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      dst_rank,
      channel,
      rank_,
      world_size_);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

static __global__ void wait_signal_kernel(
    uint32_t** signal_pads,
    int src_rank,
    int channel,
    int rank,
    int world_size) {
  if (threadIdx.x == 0) {
    acquire_signal(signal_pads[rank] + world_size * channel + src_rank);
  }
  __threadfence_system();
}

void CUDASymmetricMemory::wait_signal(int src_rank, int channel) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  wait_signal_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      src_rank,
      channel,
      rank_,
      world_size_);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

int CUDASymmetricMemory::get_rank() {
  return rank_;
}

int CUDASymmetricMemory::get_world_size() {
  return world_size_;
}

void* CUDASymmetricMemoryAllocator::alloc(
    size_t size,
    int device_idx,
    const std::string& group_name) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto driver_api = c10::cuda::DriverAPI::get();

  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  // NOLINTNEXTLINE(bugprone-signed-char-misuse)
  prop.location.id = device_idx;
  prop.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;

  size_t signal_pad_offset = at::round_up(size, 16UL);
  size_t block_size = signal_pad_offset + signal_pad_size;

  size_t granularity;
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemGetAllocationGranularity_(
      &granularity, &prop, hipMemAllocationGranularityRecommended));
  block_size = at::round_up(block_size, granularity);

  HandleType handle;
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMemCreate_(&handle, block_size, &prop, 0));

  void* ptr = nullptr;
  map_block(&ptr, handle, block_size, device_idx);

  c10::cuda::CUDAGuard guard(device_idx);
  AT_CUDA_CHECK(hipMemset(ptr, 0, block_size));

  auto block = c10::make_intrusive<Block>(
      handle, device_idx, block_size, size, signal_pad_offset, group_name);
  {
    std::unique_lock lock(mutex_);
    ptr_to_block_.emplace(ptr, std::move(block));
  }
  return ptr;
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

void CUDASymmetricMemoryAllocator::free(void* ptr) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto block = find_block(ptr);
  // Leak the cuda allocations during static deinitialization
  if (block == nullptr || is_finalizing()) {
    return;
  }
  // Initializing CUDASymmetricMemory with an allocation transfers its
  // ownership to the CUDASymmetricMemory object.
  if (block->symm_mem == nullptr) {
    auto driver_api = c10::cuda::DriverAPI::get();
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemUnmap_(
        reinterpret_cast<hipDeviceptr_t>(ptr), block->block_size));
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemRelease_(block->handle));
  }
  {
    std::unique_lock lock(mutex_);
    ptr_to_block_.erase(ptr);
  }
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

size_t CUDASymmetricMemoryAllocator::get_alloc_size(void* ptr) {
  auto block = find_block(ptr);
  TORCH_CHECK(
      block != nullptr,
      "CUDASymmetricMemoryAllocator::get_alloc_size: input must be allocated ",
      "via CUDASymmetricMemoryAllocator::alloc");
  return block->buffer_size;
}

struct RendezvousRequest {
  int device_idx;
  int pid;
  size_t block_size;
  size_t buffer_size;
  size_t signal_pad_offset;
};

void validate_rendezvous_requests(
    const std::vector<RendezvousRequest> reqs,
    int world_size) {
  TORCH_CHECK(reqs.size() == (size_t)world_size);

  std::unordered_set<int> device_indices;
  device_indices.reserve(world_size);
  for (auto req : reqs) {
    device_indices.insert(req.device_idx);
  }
  if (device_indices.size() < (size_t)world_size) {
    TORCH_CHECK(
        false,
        "CUDASymmetricMemoryAllocator::rendezvous: ",
        "detected allocations from overlapping devices ",
        "from different ranks.");
  }

  for (int r = 1; r < world_size; ++r) {
    TORCH_CHECK(reqs[r].block_size == reqs[0].block_size);
    TORCH_CHECK(reqs[r].buffer_size == reqs[0].buffer_size);
    TORCH_CHECK(reqs[r].signal_pad_offset == reqs[0].signal_pad_offset);
  }
}

c10::intrusive_ptr<SymmetricMemory> CUDASymmetricMemoryAllocator::rendezvous(
    void* ptr) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto block = find_block(ptr);
  if (block == nullptr) {
    return nullptr;
  }

  if (block->symm_mem != nullptr) {
    return block->symm_mem;
  }

  IpcChannel ipc_channel;
  auto group_info = get_group_info(block->group_name);
  auto store = group_info.store;
  int rank = group_info.rank;
  int world_size = group_info.world_size;

  auto driver_api = c10::cuda::DriverAPI::get();
  int block_fd;
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemExportToShareableHandle_(
      &block_fd, block->handle, hipMemHandleTypePosixFileDescriptor, 0));

  auto local_req = RendezvousRequest{
      .device_idx = block->device_idx,
      .pid = getpid(),
      .block_size = block->block_size,
      .buffer_size = block->buffer_size,
      .signal_pad_offset = block->signal_pad_offset};
  auto reqs = store_all_gather(store, rank, world_size, local_req);
  validate_rendezvous_requests(reqs, world_size);

  std::vector<int> pids(world_size);
  for (int r = 0; r < world_size; ++r) {
    pids[r] = reqs[r].pid;
  }
  auto imported_fds = ipc_channel.all_gather_fds(rank, pids, block_fd);

  std::vector<HandleType> handles(world_size);
  std::vector<void*> buffers(world_size, nullptr);
  std::vector<void*> signal_pads(world_size, nullptr);

  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      handles[r] = block->handle;
      buffers[r] = ptr;
      signal_pads[r] = (void*)((uintptr_t)ptr + block->signal_pad_offset);
      continue;
    }
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemImportFromShareableHandle_(
        &handles[r],
        (void*)(uintptr_t)imported_fds[r],
        hipMemHandleTypePosixFileDescriptor));
    map_block(&buffers[r], handles[r], block->block_size, block->device_idx);
    signal_pads[r] = (void*)((uintptr_t)buffers[r] + block->signal_pad_offset);
    close(imported_fds[r]);
  }
  store_barrier(store, rank, world_size);
  close(block_fd);

  hipMemGenericAllocationHandle_t mc_handle{};
  void* mc_addr = nullptr;
#if defined(CUDART_SUPPORTS_MULTICAST)
  // We have to further check if the driver supports multicast
  if (has_multicast_support()) {
    // Rank 0 creates a multicast object and share it with peers
    if (rank == 0) {
      CUmulticastObjectProp mc_prop{};
      mc_prop.numDevices = world_size;
      mc_prop.handleTypes = hipMemHandleTypePosixFileDescriptor;
      mc_prop.size = block->block_size;

      hipError_t res = driver_api->cuMulticastCreate_(&mc_handle, &mc_prop);
      TORCH_CHECK(res == hipSuccess);

      int mc_fd;
      C10_CUDA_DRIVER_CHECK(driver_api->cuMemExportToShareableHandle_(
          &mc_fd, mc_handle, hipMemHandleTypePosixFileDescriptor, 0));
      ipc_channel.broadcast_fds(rank, 0, pids, mc_fd);
      // Ref count is incremented as soon as SCM_RIGHTS send happens
      close(mc_fd);
    } else {
      int mc_fd = ipc_channel.broadcast_fds(rank, 0, pids, -1);
      C10_CUDA_DRIVER_CHECK(driver_api->cuMemImportFromShareableHandle_(
          &mc_handle,
          (void*)(uintptr_t)mc_fd,
          hipMemHandleTypePosixFileDescriptor));
      close(mc_fd);
    }
    // All rank adds their physical allocation to the multicast object
    C10_CUDA_DRIVER_CHECK(
        driver_api->cuMulticastAddDevice_(mc_handle, block->device_idx));
    C10_CUDA_DRIVER_CHECK(driver_api->cuMulticastBindMem_(
        mc_handle, 0, block->handle, 0, block->block_size, 0));

    map_block(&mc_addr, mc_handle, block->block_size, block->device_idx);
    store_barrier(store, rank, world_size);
  }
#endif

  // Initializing CUDASymmetricMemory with an allocation transfers its
  // ownership to the CUDASymmetricMemory object. So that outstanding
  // references to the CUDASymmetricMemory object can keep the allocation
  // alive.
  block->symm_mem = c10::make_intrusive<CUDASymmetricMemory>(
      std::move(handles),
      block->block_size,
      std::move(buffers),
      std::move(signal_pads),
      mc_handle,
      mc_addr,
      block->buffer_size,
      block->device_idx,
      group_info.rank,
      group_info.world_size);
  return block->symm_mem;
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

bool CUDASymmetricMemoryAllocator::is_rendezvous_completed(void* ptr) {
  auto block = find_block(ptr);
  TORCH_CHECK(
      block != nullptr,
      "CUDASymmetricMemoryAllocator::is_rendezvous_completed: input must be allocated ",
      "via CUDASymmetricMemoryAllocator::alloc");
  return block->symm_mem != nullptr;
}

bool CUDASymmetricMemoryAllocator::has_multicast_support() {
  return ::has_multicast_support();
}

c10::intrusive_ptr<Block> CUDASymmetricMemoryAllocator::find_block(void* ptr) {
  std::shared_lock lock(mutex_);
  auto it = ptr_to_block_.find(ptr);
  if (it == ptr_to_block_.end()) {
    return nullptr;
  }
  return it->second;
}

struct RegisterCUDASymmetricMemoryAllocator {
  RegisterCUDASymmetricMemoryAllocator() {
    register_allocator(
        c10::DeviceType::CUDA,
        c10::make_intrusive<CUDASymmetricMemoryAllocator>());
  }
};

static RegisterCUDASymmetricMemoryAllocator register_allocator_;

} // namespace symmetric_memory
} // namespace c10d
